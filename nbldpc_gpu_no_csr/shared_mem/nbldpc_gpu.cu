#include "hip/hip_runtime.h"

#include "stdio.h"
#include "stdlib.h"
#include <math.h>
#include <time.h>
#include <string.h>
//#include <ap_fixed.h>
#include "nbldpc.h"
#include <hip/hip_runtime.h>      // CUDA Runtime Functions
#include <hip/hip_runtime_api.h>

extern "C" {
    #include "nbldpc_gpu.h"
}


#ifdef VS
#define _CRT_SECURE_NO_DEPRECATE
#endif

#if Q==4
__constant__ unsigned short c_col_row[3][2];
__constant__ unsigned short c_row_col[1][6];
__constant__ unsigned char c_h_nb[3][6];
#elif Q==8
__constant__ unsigned short c_col_row[14][3];
__constant__ unsigned short c_row_col[1][42];
__constant__ unsigned char c_h_nb[14][42];
#elif Q==16
__constant__ unsigned short c_col_row[150][6];
__constant__ unsigned short c_row_col[3][300];
__constant__ unsigned char c_h_nb[150][300];
#elif Q==32
__constant__ unsigned short c_col_row[310][6];
__constant__ unsigned short c_row_col[3][620];
#endif


__constant__ unsigned short c_M;
__constant__ unsigned short c_N;
__constant__ unsigned char c_add[Q][Q];
__constant__ unsigned char c_mult[Q][Q];
__constant__ unsigned char c_inv[Q];
__constant__ unsigned char c_w_row;
__constant__ unsigned char c_w_col;

extern unsigned char *h_F;
extern unsigned char *h_B;

#if Q==32
	//===================================
	// CUDA kernel
	//===================================
	__global__ void GPU_FB_metrics( unsigned char * d_alpha,  unsigned char * d_F,  unsigned char * d_B, int iter, unsigned char * d_h_nb){
		
		unsigned short x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned short z=threadIdx.z+blockIdx.z*blockDim.z;
		unsigned char min_F, max_F, min_B, max_B, a_F, a_B;

		__shared__ unsigned char s_alpha[32][6][32];
		__shared__ unsigned char s_F[32][32][6];
		__shared__ unsigned char s_B[32][32][6];

		if(x<c_M){


			for(unsigned char index=0; index<c_w_row; index++){
				s_alpha[threadIdx.x][index][z]=d_alpha[(Q*c_N*x)+(Q*c_col_row[x][index])+z];
			}
			__syncthreads();

			s_F[threadIdx.x][z][0]=s_alpha[threadIdx.x][0][c_mult[c_inv[d_h_nb[(x*c_N)+c_col_row[x][0]]]][z]];
			//if(x==0)
			//printf("F[%d][%d][0]=%d\n",x,z,s_F[threadIdx.x][z][0]);

			s_B[threadIdx.x][z][c_w_row-1]=s_alpha[threadIdx.x][c_w_row-1][c_mult[c_inv[d_h_nb[(x*c_N)+c_col_row[x][c_w_row-1]]]][z]];
			//if(x==1)
			//printf(" B[%d][%d][%d]=%d, row=%d, lastcol=%d, mult=%d\n",x ,z,c_w_row-1, s_B[threadIdx.x][z][c_w_row-1], x, c_col_row[x][c_w_row-1],c_mult[c_inv[c_h_nb[x][c_col_row[x][0]]]][z]);

			
			d_F[(x*Q*c_w_row)+z*c_w_row]=s_F[threadIdx.x][z][0];
			//if(x==0)
			//printf("F[%d][%d]=%d\n",x,z,d_F[(x*Q*c_w_row)+z*c_w_row]);

			d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row-1]=s_B[threadIdx.x][z][c_w_row-1];

			__syncthreads();
			


			for(unsigned char index=1; index<c_w_row; index++){
				//s_alpha[threadIdx.x][0][z]=d_alpha[(Q*c_N*x)+(Q*c_col_row[x][index])+z];
				//s_alpha[threadIdx.x][1][z]=d_alpha[(Q*c_N*x)+(Q*c_col_row[x][c_w_row - index -1])+z];
				min_F = (s_F[threadIdx.x][z][index-1] < s_alpha[threadIdx.x][index][0]) ? s_alpha[threadIdx.x][index][0] : s_F[threadIdx.x][z][index-1];
				//if(x==0 && index==1)
				//printf("min_F=%d, F[%d][%d]=%d, ALPHAmn_[%d][%d][0]=%d\n", min_F,z,index, s_F[threadIdx.x][z][index-1],x,c_col_row[x][index], s_alpha[threadIdx.x][index][z]);
				min_B = (s_B[threadIdx.x][z][c_w_row - index] < s_alpha[threadIdx.x][c_w_row - index-1][0]) ? s_alpha[threadIdx.x][c_w_row - index-1][0] : s_B[threadIdx.x][z][c_w_row - index];
				//if(x==1)
				//printf("min_F=%d, B[%d][%d]=%d, ALPHAmn_[%d][%d][0]=%d\n", min_B,z,c_w_row - index, d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row - index],x,c_col_row[x][c_w_row - index -1], d_alpha[(Q*c_N*x)+(Q*c_col_row[x][c_w_row - index -1])]);
				for(unsigned char b=1; b<Q; b++){
					a_F=c_add[z][c_mult[d_h_nb[(x*c_N)+c_col_row[x][index]]][b]];
					a_B=c_add[z][c_mult[d_h_nb[(x*c_N)+c_col_row[x][c_w_row - index -1]]][b]];
					
					
					max_F = (s_F[threadIdx.x][a_F][index-1] < s_alpha[threadIdx.x][index][b]) ? s_alpha[threadIdx.x][index][b] : s_F[threadIdx.x][a_F][index-1];
					max_B = (s_B[threadIdx.x][a_B][c_w_row - index] < s_alpha[threadIdx.x][c_w_row - index-1][b]) ? s_alpha[threadIdx.x][c_w_row - index-1][b] : s_B[threadIdx.x][a_B][c_w_row - index] ;
					//if(x==0 && index==2)
					//printf("min_F=%d, max_F=%d, F[%d][%d][%d]=%d,alpha[%d][%d][%d]=%d \n", min_F,max_F,x,a_F,index-1,d_F[(x*Q*c_w_row)+(a_F*c_w_row)+index-1], x,c_col_row[x][index],b, d_alpha[(Q*c_N*x)+(Q*c_col_row[x][index])+b]);

					min_F = (min_F < max_F) ? min_F : max_F;
					min_B = (min_B < max_B) ? min_B : max_B;

					//if(x==0 && index==2)
					//printf("min_F=%d, max_F=%d, F[%d][%d][%d]=%d,alpha[%d][%d][%d]=%d \n", min_F,max_F,x,a_F,index-1,s_F[threadIdx.x][a_F][index-1], x,c_col_row[x][index],b, s_alpha[threadIdx.x][0][b]);
					
				}
				__syncthreads();

				d_F[(x*Q*c_w_row)+(z*c_w_row)+index]=min_F;
				s_F[threadIdx.x][z][index]=min_F;
				//if(x==0 && index==2)
				//printf("F[%d][%d][%d]=%d\n",x,z,index,d_F[(x*Q*c_w_row)+(z*c_w_row)+index]);
				d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row - index -1]=min_B;
				s_B[threadIdx.x][z][c_w_row - index-1]=min_B;
				//if(x==7)
				//printf("B[%d][%d][%d]=%d\n",x,z,c_w_row - index -1,d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row - index -1]);
				//__syncthreads();
			}
		}

		

	} 

	__global__ void GPU_CN( unsigned char * d_beta,  unsigned char * d_F,  unsigned char * d_B, int iter, unsigned char * d_h_nb){
		unsigned int x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned int y=threadIdx.y+blockIdx.y*blockDim.y;
		unsigned int z=threadIdx.z+blockIdx.z*blockDim.z;

		__shared__ unsigned char s_F[5][32][6];
		__shared__ unsigned char s_B[5][32][6];

		s_F[threadIdx.x][z][y]=d_F[(x*Q*c_w_row)+(z*c_w_row)+y];
		s_B[threadIdx.x][z][y]=d_B[(x*Q*c_w_row)+(z*c_w_row)+y];
		__syncthreads();

		if(y==0){
			d_beta[(x*c_N*Q)+(Q*c_col_row[x][0])+z]=s_B[threadIdx.x][c_mult[d_h_nb[(x*c_N)+c_col_row[x][0]]][z]][1];
			//if(x==1)
			//printf("BETAmn_[%d][%d][%d]=%d\n", x, c_col_row[x][0], z, d_beta[(x*c_N*Q)+(Q*c_col_row[x][0])+z]);

		}
		else if(y==c_w_row-1){
			d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z]=s_F[threadIdx.x][c_mult[d_h_nb[(x*c_N)+c_col_row[x][y]]][z]][y-1];
			//if(x==0)
			//printf("BETAmn_[%d][%d][%d]=%d\n", x, c_col_row[x][y], z, d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z]);

		}
		else{
			unsigned char min, max, a;
			min = (s_F[threadIdx.x][c_mult[d_h_nb[(x*c_N)+c_col_row[x][y]]][z]][y-1] < s_B[threadIdx.x][0][y+1]) ? s_B[threadIdx.x][0][y+1] : s_F[threadIdx.x][c_mult[d_h_nb[(x*c_N)+c_col_row[x][y]]][z]][y-1];
			//if(x==0 && z==2) 
			//printf("mmin=%d F[%d][%d][%d]=%d, B[%d][%d][%d]=%d, col=%d\n", min,x, (c_mult[c_h_nb[x][c_col_row[x][y]]][z]), y-1, d_F[(x*Q*c_w_row)+((c_mult[c_h_nb[x][c_col_row[x][y]]][z])*c_w_row)+y-1], x, 0, y+1, d_B[(x*Q*c_w_row)+y+1],c_col_row[x][y]);
			for(unsigned char b=1; b<Q; b++){
				a=c_add[b][c_mult[d_h_nb[(x*c_N)+c_col_row[x][y]]][z]];
				max = (s_F[threadIdx.x][a][y-1] < s_B[threadIdx.x][b][y+1]) ? s_B[threadIdx.x][b][y+1] : s_F[threadIdx.x][a][y-1] ;
				//if(x==0 && z==2)
				//printf("max=%d min=%d z=%d, b=%d, a=%d\n", max, min, z, b, a);

				min = (min < max) ? min : max;
				//if(x==0 && z==2)
				//printf("min=%d\n", min);

			}
			d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z]=min;
			//if(x==0)
			//printf("BETAmn_[%d][%d][%d]=%d y=%d\n", x, c_col_row[x][y], z, d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z], y);

		}

	}

	__global__ void GPU_VN( unsigned char * d_beta,  unsigned char * d_alpha,  unsigned char * d_alpha_t, unsigned char * d_alpha_t2, unsigned char * d_gamma, int iter){
		unsigned int x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned int y=threadIdx.y+blockIdx.y*blockDim.y;
		unsigned int z=threadIdx.z+blockIdx.z*blockDim.z;
		unsigned char temp=0;

		__shared__ unsigned char s_beta[3][10][32];
		__shared__ unsigned char s_alpha_t[3][10][32];
		__shared__ unsigned char s_alpha_t2[3][10][32];

		s_beta[x][threadIdx.y][z]=d_beta[(c_row_col[x][y]*c_N*Q)+(y*Q)+z];
			__syncthreads();

			for(unsigned char index=0; index<c_w_col; index++){
				if(index!=x){
					temp=temp+s_beta[index][threadIdx.y][z];
				}
				__syncthreads();
			}

			//if( c_row_col[x][y]==0 && y==8 )
			//printf("temp=%d, GAMMAn_[%d][%d]=%d\n", temp, y,z,d_gamma[(y*Q)+z]);


			//__syncthreads();

			s_alpha_t[x][threadIdx.y][z]= temp+ d_gamma[(y*Q)+z];
			
			s_alpha_t2[x][threadIdx.y][z]=z;
			__syncthreads();

			//if(x==0 && y==0)
			//printf("alpha_t[%d][%d][%d]=%d\n",x,y,z,d_alpha_t[(x*c_N*Q)+(Q*y)+z]);


			
			for(unsigned char stride=blockDim.z/2; stride>0; stride>>=1){
				if(z<stride){
					/* if( c_row_col[x][y]==0 ){
					printf("t=%d min=%d y=%d z=%d\n", d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z]], d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride]],y,z);
					} */
					s_alpha_t2[x][threadIdx.y][z]= (s_alpha_t[x][threadIdx.y][s_alpha_t2[x][threadIdx.y][z]] > s_alpha_t[x][threadIdx.y][s_alpha_t2[x][threadIdx.y][z+stride]]) ? s_alpha_t2[x][threadIdx.y][z+stride] : s_alpha_t2[x][threadIdx.y][z];
					//if( c_row_col[x][y]==0 ){
						//printf("t=%d min=%d y=%d z=%d\n", d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z]], d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride]],y,z);
						//}
					//if(x==0 && y==0 )
					//printf("alpha_t2[%d][%d][%d]=%d\n",x,y,z,d_alpha_t2[(x*c_N*Q)+(Q*y)+z]);

				}
				__syncthreads();
			}

			//__syncthreads();

			
			//if(x==0  && z==0)
			//printf("alpha_t2[%d][%d][%d]=%d\n",x,y,z,d_alpha_t2[(x*c_N*Q)+(Q*y)+z]);

			//if(c_row_col[x][y]==41 )
			//printf("ALPHA_t[%d]=%d ALPHA_t2[%d]=%d\n", z, d_alpha_t[(x*c_N*Q)+(Q*y)+z], d_alpha_t2[(x*c_N*Q)+(Q*y)],  d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)]]);
			
			temp=s_alpha_t[x][threadIdx.y][s_alpha_t2[x][threadIdx.y][0]];
			//__syncthreads();
			//if( c_row_col[x][y]==0 && y==155 && z==1)
			//printf("temp=%d\n", temp);
			d_alpha[(c_row_col[x][y]*c_N*Q)+(y*Q)+z] = s_alpha_t[x][threadIdx.y][z] - temp;

	}

#else
	//===================================
	// CUDA kernel
	//===================================
	__global__ void GPU_FB_metrics( unsigned char * d_alpha,  unsigned char * d_F,  unsigned char * d_B, int iter){
		
		unsigned short x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned short z=threadIdx.z+blockIdx.z*blockDim.z;
		unsigned char min_F, max_F, min_B, max_B, a_F, a_B;
		
		#if Q==4
		__shared__ unsigned char s_alpha[3][2][4];
		__shared__ unsigned char s_F[3][4][2];
		__shared__ unsigned char s_B[3][4][2];
		#elif Q==8
		__shared__ unsigned char s_alpha[14][2][8];
		__shared__ unsigned char s_F[14][8][2];
		__shared__ unsigned char s_B[14][8][2];
		#elif Q==16
		__shared__ unsigned char s_alpha[64][6][16];
		__shared__ unsigned char s_F[64][16][6];
		__shared__ unsigned char s_B[64][16][6];
		#endif

		/* if(x==2)
		printf("alpha[%d][0][%d]=%d\n",x,z,d_alpha[(Q*c_N*x)+(Q*c_col_row[x][0])+z]); */


		if(x<c_M){


			for(unsigned char index=0; index<c_w_row; index++){
				s_alpha[threadIdx.x][index][z]=d_alpha[(Q*c_N*x)+(Q*c_col_row[x][index])+z];
			}
			__syncthreads();

			s_F[threadIdx.x][z][0]=s_alpha[threadIdx.x][0][c_mult[c_inv[c_h_nb[x][c_col_row[x][0]]]][z]];
			/* if(x==2)
			printf("F[%d][%d][0]=%d\n",x,z,s_F[threadIdx.x][z][0]); */

			s_B[threadIdx.x][z][c_w_row-1]=s_alpha[threadIdx.x][c_w_row-1][c_mult[c_inv[c_h_nb[x][c_col_row[x][c_w_row-1]]]][z]];
			//if(x==2)
			//printf(" B[%d][%d][%d]=%d, row=%d, lastcol=%d, mult=%d\n",x ,z,c_w_row-1, s_B[threadIdx.x][z][c_w_row-1], x, c_col_row[x][c_w_row-1],c_mult[c_inv[c_h_nb[x][c_col_row[x][0]]]][z]);

			
			d_F[(x*Q*c_w_row)+z*c_w_row]=s_F[threadIdx.x][z][0];
			//if(x==0)
			//printf("F[%d][%d]=%d\n",x,z,d_F[(x*Q*c_w_row)+z*c_w_row]);

			d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row-1]=s_B[threadIdx.x][z][c_w_row-1];

			__syncthreads();
			


			for(unsigned char index=1; index<c_w_row; index++){
				//s_alpha[threadIdx.x][0][z]=d_alpha[(Q*c_N*x)+(Q*c_col_row[x][index])+z];
				//s_alpha[threadIdx.x][1][z]=d_alpha[(Q*c_N*x)+(Q*c_col_row[x][c_w_row - index -1])+z];
				min_F = (s_F[threadIdx.x][z][index-1] < s_alpha[threadIdx.x][index][0]) ? s_alpha[threadIdx.x][index][0] : s_F[threadIdx.x][z][index-1];
				//if(x==0 && index==1)
				//printf("min_F=%d, F[%d][%d]=%d, ALPHAmn_[%d][%d][0]=%d\n", min_F,z,index, s_F[threadIdx.x][z][index-1],x,c_col_row[x][index], s_alpha[threadIdx.x][index][z]);
				min_B = (s_B[threadIdx.x][z][c_w_row - index] < s_alpha[threadIdx.x][c_w_row - index-1][0]) ? s_alpha[threadIdx.x][c_w_row - index-1][0] : s_B[threadIdx.x][z][c_w_row - index];
				//if(x==1)
				//printf("min_F=%d, B[%d][%d]=%d, ALPHAmn_[%d][%d][0]=%d\n", min_B,z,c_w_row - index, d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row - index],x,c_col_row[x][c_w_row - index -1], d_alpha[(Q*c_N*x)+(Q*c_col_row[x][c_w_row - index -1])]);
				for(unsigned char b=1; b<Q; b++){
					a_F=c_add[z][c_mult[c_h_nb[x][c_col_row[x][index]]][b]];
					a_B=c_add[z][c_mult[c_h_nb[x][c_col_row[x][c_w_row - index -1]]][b]];
					//if(x==0)
					//printf("a[%d][%d][%d][%d]=%d\n",x, index, z, b, a_B);
					
					
					max_F = (s_F[threadIdx.x][a_F][index-1] < s_alpha[threadIdx.x][index][b]) ? s_alpha[threadIdx.x][index][b] : s_F[threadIdx.x][a_F][index-1];
					max_B = (s_B[threadIdx.x][a_B][c_w_row - index] < s_alpha[threadIdx.x][c_w_row - index-1][b]) ? s_alpha[threadIdx.x][c_w_row - index-1][b] : s_B[threadIdx.x][a_B][c_w_row - index] ;
					//if(x==0 && index==2)
					//printf("min_F=%d, max_F=%d, F[%d][%d][%d]=%d,alpha[%d][%d][%d]=%d \n", min_F,max_F,x,a_F,index-1,d_F[(x*Q*c_w_row)+(a_F*c_w_row)+index-1], x,c_col_row[x][index],b, d_alpha[(Q*c_N*x)+(Q*c_col_row[x][index])+b]);

					//if(x==0)
					//printf("max_B[%d][%d][%d]=%d\n",z,x, b, max_B);
					

					min_F = (min_F < max_F) ? min_F : max_F;
					min_B = (min_B < max_B) ? min_B : max_B;

					/* if(x==0)
					printf("min_B[%d][%d][%d]=%d\n",z,x, b, min_B); */

					//if(x==0 && index==2)
					//printf("min_F=%d, max_F=%d, F[%d][%d][%d]=%d,alpha[%d][%d][%d]=%d \n", min_F,max_F,x,a_F,index-1,s_F[threadIdx.x][a_F][index-1], x,c_col_row[x][index],b, s_alpha[threadIdx.x][0][b]);
					
				}
				__syncthreads();

				d_F[(x*Q*c_w_row)+(z*c_w_row)+index]=min_F;
				s_F[threadIdx.x][z][index]=min_F;

				//if(x==0)
				//printf("min_B[%d][%d]=%d\n",z,x, d_F[(x*Q*c_w_row)+(z*c_w_row)+index]);
				//if(x==0 && index==2)
				//printf("F[%d][%d][%d]=%d\n",x,z,index,d_F[(x*Q*c_w_row)+(z*c_w_row)+index]);
				d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row - index -1]=min_B;
				s_B[threadIdx.x][z][c_w_row - index-1]=min_B;

				//if(x==0)
				//printf("min_B[%d][%d]=%d\n",z,x, d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row - index -1]);

				//if(x==7)
				//printf("B[%d][%d][%d]=%d\n",x,z,c_w_row - index -1,d_B[(x*Q*c_w_row)+(z*c_w_row)+c_w_row - index -1]);
				//__syncthreads();
			}
		} 

	} 

	__global__ void GPU_CN( unsigned char * d_beta,  unsigned char * d_F,  unsigned char * d_B, int iter){
		unsigned int x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned int y=threadIdx.y+blockIdx.y*blockDim.y;
		unsigned int z=threadIdx.z+blockIdx.z*blockDim.z;
		

		#if Q==4
		__shared__ unsigned char s_F[3][4][2];
		__shared__ unsigned char s_B[3][4][2];
		#elif Q==8
		__shared__ unsigned char s_F[14][8][2];
		__shared__ unsigned char s_B[14][8][2];
		#elif Q==16
		__shared__ unsigned char s_F[10][16][6];
		__shared__ unsigned char s_B[10][16][6];
		#endif

		//if(x<c_M){

			s_F[threadIdx.x][z][y]=d_F[(x*Q*c_w_row)+(z*c_w_row)+y];
			s_B[threadIdx.x][z][y]=d_B[(x*Q*c_w_row)+(z*c_w_row)+y];
			__syncthreads();

			if(y==0){
				d_beta[(x*c_N*Q)+(Q*c_col_row[x][0])+z]=s_B[threadIdx.x][c_mult[c_h_nb[x][c_col_row[x][0]]][z]][1];
				if(x==1)
				printf("BETAmn_[%d][%d][%d]=%d, new_Z=%d\n", x, c_col_row[x][0], z, d_beta[(x*c_N*Q)+(Q*c_col_row[x][0])+z], c_mult[c_h_nb[x][c_col_row[x][0]]][z]);

			}
			else if(y==c_w_row-1){
				d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z]=s_F[threadIdx.x][c_mult[c_h_nb[x][c_col_row[x][y]]][z]][y-1];
				//if(x==0)
				//printf("BETAmn_[%d][%d][%d]=%d\n", x, c_col_row[x][y], z, d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z]);

			}
			else{
				unsigned char min, max, a;
				min = (s_F[threadIdx.x][c_mult[c_h_nb[x][c_col_row[x][y]]][z]][y-1] < s_B[threadIdx.x][0][y+1]) ? s_B[threadIdx.x][0][y+1] : s_F[threadIdx.x][c_mult[c_h_nb[x][c_col_row[x][y]]][z]][y-1];
				//if(x==0 && z==2) 
				//printf("mmin=%d F[%d][%d][%d]=%d, B[%d][%d][%d]=%d, col=%d\n", min,x, (c_mult[c_h_nb[x][c_col_row[x][y]]][z]), y-1, d_F[(x*Q*c_w_row)+((c_mult[c_h_nb[x][c_col_row[x][y]]][z])*c_w_row)+y-1], x, 0, y+1, d_B[(x*Q*c_w_row)+y+1],c_col_row[x][y]);
				for(unsigned char b=1; b<Q; b++){
					a=c_add[b][c_mult[c_h_nb[x][c_col_row[x][y]]][z]];
					max = (s_F[threadIdx.x][a][y-1] < s_B[threadIdx.x][b][y+1]) ? s_B[threadIdx.x][b][y+1] : s_F[threadIdx.x][a][y-1] ;
					//if(x==0 && z==2)
					//printf("max=%d min=%d z=%d, b=%d, a=%d\n", max, min, z, b, a);

					min = (min < max) ? min : max;
					//if(x==0 && z==2)
					//printf("min=%d\n", min);

				}
				d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z]=min;
				//if(x==0)
				//printf("BETAmn_[%d][%d][%d]=%d y=%d\n", x, c_col_row[x][y], z, d_beta[(x*c_N*Q)+(Q*c_col_row[x][y])+z], y);

			}
		//}

	}

	__global__ void GPU_VN( unsigned char * d_beta,  unsigned char * d_alpha,  unsigned char * d_alpha_t, unsigned char * d_alpha_t2, unsigned char * d_gamma, int iter){
		unsigned int x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned int y=threadIdx.y+blockIdx.y*blockDim.y;
		unsigned int z=threadIdx.z+blockIdx.z*blockDim.z;
		unsigned short temp=0;

		#if Q==4
		__shared__ unsigned char s_beta[1][6][4];
		__shared__ unsigned char s_alpha_t[1][6][4];
		__shared__ unsigned char s_alpha_t2[1][6][4];
		#elif Q==8
		__shared__ unsigned char s_beta[1][42][8];
		__shared__ unsigned char s_alpha_t[1][42][8];
		__shared__ unsigned char s_alpha_t2[1][42][8];
		#elif Q==16
		__shared__ unsigned char s_beta[3][21][16];
		__shared__ unsigned char s_alpha_t[3][21][16];
		__shared__ unsigned char s_alpha_t2[3][21][16];
		#endif

		if(y<c_N){

			s_beta[x][threadIdx.y][z]=d_beta[(c_row_col[x][y]*c_N*Q)+(y*Q)+z];
			__syncthreads();

			for(unsigned char index=0; index<c_w_col; index++){
				if(index!=x){
					temp=temp+s_beta[index][threadIdx.y][z];
				}
				__syncthreads();
			}

			//if( c_row_col[x][y]==0 && y==8 )
			//printf("temp=%d, GAMMAn_[%d][%d]=%d\n", temp, y,z,d_gamma[(y*Q)+z]);


			//__syncthreads();

			s_alpha_t[x][threadIdx.y][z]= temp+ d_gamma[(y*Q)+z];
			
			s_alpha_t2[x][threadIdx.y][z]=z;
			__syncthreads();

			//if(x==0 && y==0)
			//printf("alpha_t[%d][%d][%d]=%d\n",x,y,z,d_alpha_t[(x*c_N*Q)+(Q*y)+z]);


			
			for(unsigned char stride=blockDim.z/2; stride>0; stride>>=1){
				if(z<stride){
					/* if( c_row_col[x][y]==0 ){
					printf("t=%d min=%d y=%d z=%d\n", d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z]], d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride]],y,z);
					} */
					s_alpha_t2[x][threadIdx.y][z]= (s_alpha_t[x][threadIdx.y][s_alpha_t2[x][threadIdx.y][z]] > s_alpha_t[x][threadIdx.y][s_alpha_t2[x][threadIdx.y][z+stride]]) ? s_alpha_t2[x][threadIdx.y][z+stride] : s_alpha_t2[x][threadIdx.y][z];
					//if( c_row_col[x][y]==0 ){
						//printf("t=%d min=%d y=%d z=%d\n", d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z]], d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride]],y,z);
						//}
					//if(x==0 && y==0 )
					//printf("alpha_t2[%d][%d][%d]=%d\n",x,y,z,d_alpha_t2[(x*c_N*Q)+(Q*y)+z]);

				}
				__syncthreads();
			}

			//__syncthreads();

			
			//if(x==0  && z==0)
			//printf("alpha_t2[%d][%d][%d]=%d\n",x,y,z,d_alpha_t2[(x*c_N*Q)+(Q*y)+z]);

			//if(c_row_col[x][y]==41 )
			//printf("ALPHA_t[%d]=%d ALPHA_t2[%d]=%d\n", z, d_alpha_t[(x*c_N*Q)+(Q*y)+z], d_alpha_t2[(x*c_N*Q)+(Q*y)],  d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)]]);
			
			temp=s_alpha_t[x][threadIdx.y][s_alpha_t2[x][threadIdx.y][0]];
			//__syncthreads();
			//if( c_row_col[x][y]==0 && y==155 && z==1)
			//printf("temp=%d\n", temp);
			d_alpha[(c_row_col[x][y]*c_N*Q)+(y*Q)+z] = s_alpha_t[x][threadIdx.y][z] - temp;
			
			//__syncthreads();
			
			//if( c_row_col[x][y]==0 && y==155 && z==1)
			//printf("alpha[%d][%d][%d]=%d\n",c_row_col[x][y],y,z,d_alpha[(c_row_col[x][y]*c_N*Q)+(y*Q)+z]);

			
			//if(y==0)
			//printf("F[%d][0]=%d, B[%d][%d]=%d, row=%d, lastcol=%d, mult=%d\n", z, d_F[(x*Q*c_N)+z*c_N],z,c_w_row-1, d_B[(x*Q*c_N)+(z*c_N)+c_w_row-1], x, c_col_row[x][c_w_row-1],c_mult[c_inv[c_h_nb[x][c_col_row[x][0]]]][z]);
		} 


			/* for(unsigned char index=0; index<c_w_col; index++){
				if(index!=x){
					temp=temp+d_beta[(c_row_col[index][y]*c_N*Q)+(y*Q)+z];
				}
				__syncthreads();
				//if( c_row_col[x][y]==0 && y==155 && z==1)
				//printf("temp=%d\n",temp);

			}

			
			
			__syncthreads();

			d_alpha_t[(x*c_N*Q)+(Q*y)+z]= temp+ d_gamma[(y*Q)+z];
			__syncthreads();
			//if( c_row_col[x][y]==0 && y==155 && z==1)
			//printf("temp=%d, GAMMAn_[%d][%d]=%d\n", temp, y,z,d_gamma[(y*Q)+z]);

			d_alpha_t2[(x*c_N*Q)+(Q*y)+z]=z;
			__syncthreads();

			//if(x==0 && y==0)
			//printf("alpha_t[%d][%d][%d]=%d\n",x,y,z,d_alpha_t[(x*c_N*Q)+(Q*y)+z]);


			for(unsigned char stride=1; stride<Q; stride*=2){
				if(z%(2*stride)==0){
					/* if( c_row_col[x][y]==0 ){
					printf("t=%d min=%d y=%d z=%d\n", d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z]], d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride]],y,z);
					} */
					/*d_alpha_t2[(x*c_N*Q)+(Q*y)+z]= (d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z]] > d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride]]) ? d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride] : d_alpha_t2[(x*c_N*Q)+(Q*y)+z];
					//if( c_row_col[x][y]==0 ){
						//printf("t=%d min=%d y=%d z=%d\n", d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z]], d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)+z+stride]],y,z);
						//}
					//if(x==0 && y==0 )
					//printf("alpha_t2[%d][%d][%d]=%d\n",x,y,z,d_alpha_t2[(x*c_N*Q)+(Q*y)+z]);

				}
				__syncthreads();
			}

			__syncthreads();
			

			//if(x==0  && z==0)
			//printf("alpha_t2[%d][%d][%d]=%d\n",x,y,z,d_alpha_t2[(x*c_N*Q)+(Q*y)+z]);

			//if(c_row_col[x][y]==41 )
				//printf("ALPHA_t[%d]=%d ALPHA_t2[%d]=%d\n", z, d_alpha_t[(x*c_N*Q)+(Q*y)+z], d_alpha_t2[(x*c_N*Q)+(Q*y)],  d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)]]);
				
			temp=d_alpha_t[(x*c_N*Q)+(Q*y)+d_alpha_t2[(x*c_N*Q)+(Q*y)]];
			__syncthreads();
			//if( c_row_col[x][y]==0 && y==155 && z==1)
			//printf("temp=%d\n", temp);
			d_alpha[(c_row_col[x][y]*c_N*Q)+(y*Q)+z] = d_alpha_t[(x*c_N*Q)+(Q*y)+z] - temp;
			
			__syncthreads();
			
			//if( c_row_col[x][y]==0 && y==155 && z==1)
			//printf("alpha[%d][%d][%d]=%d\n",c_row_col[x][y],y,z,d_alpha[(c_row_col[x][y]*c_N*Q)+(y*Q)+z]);

			
			//if(y==0)
			//printf("F[%d][0]=%d, B[%d][%d]=%d, row=%d, lastcol=%d, mult=%d\n", z, d_F[(x*Q*c_N)+z*c_N],z,c_w_row-1, d_B[(x*Q*c_N)+(z*c_N)+c_w_row-1], x, c_col_row[x][c_w_row-1],c_mult[c_inv[c_h_nb[x][c_col_row[x][0]]]][z]);
		} */

	}
#endif


//===================================
// CUDA Wrapper
//===================================

extern "C" int cuda_minmax(unsigned char* h_beta, unsigned char* ALPHAmn_, unsigned char* GAMMAn_, unsigned char* h_F,unsigned char* h_B,int* iteration, int* decoded_bit){
	
	#if Q==4
		const unsigned char add[4][4] = {{ 0,1,2,3 }, { 1,0,3,2 },{ 2,3,0,1 },{ 3,2,1,0 }};
		const unsigned char mult[4][4] = {{ 0,0,0,0}, { 0,1,2,3 },{ 0,2,3,1 },{ 0,3,1,2 }};
		const unsigned char inv[4] = { 0,1,3,2};
	#elif Q==8
		const unsigned char add[8][8] = {{ 0,1,2,3,4,5,6,7 }, { 1,0,3,2,5,4,7,6 },{ 2,3,0,1,6,7,4,5 },{ 3,2,1,0,7,6,5,4 },{ 4,5,6,7,0,1,2,3 }, { 5,4,7,6,1,0,3,2 },{ 6,7,4,5,2,3,0,1 },{ 7,6,5,4,3,2,1,0 }};
		const unsigned char mult[8][8] = {{ 0,0,0,0,0,0,0,0}, { 0,1,2,3,4,5,6,7 },{ 0,2,4,6,3,1,7,5 },{ 0,3,6,5,7,4,1,2 },{ 0,4,3,7,6,2,5,1}, { 0,5,1,4,2,7,3,6 },{ 0,6,7,1,5,3,2,4 },{ 0,7,5,2,1,6,4,3 }};
		const unsigned char inv[8] = { 0,1,5,6,7,2,3,4};
	#elif Q==16
		const unsigned char add[16][16] = {	{ 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15},
											{ 1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14 },
											{ 2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13 },
											{ 3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12 },
											{ 4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11 },
											{ 5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10 },
											{ 6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9 },
											{ 7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8 },
											{ 8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7 },
											{ 9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6 },
											{ 10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5 },
											{ 11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4 },
											{ 12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3 },
											{ 13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2 },
											{ 14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1 },
											{ 15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0 }};

		const unsigned char mult[16][16] = {{ 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
											{ 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15 },
											{ 0,2,4,6,8,10,12,14,3,1,7,5,11,9,15,13 },
											{ 0,3,6,5,12,15,10,9,11,8,13,14,7,4,1,2 },
											{ 0,4,8,12,3,7,11,15,6,2,14,10,5,1,13,9},
											{ 0,5,10,15,7,2,13,8,14,11,4,1,9,12,3,6},
											{ 0,6,12,10,11,13,7,1,5,3,9,15,14,8,2,4},
											{ 0,7,14,9,15,8,1,6,13,10,3,4,2,5,12,11},
											{ 0,8,3,11,6,14,5,13,12,4,15,7,10,2,9,1},
											{ 0,9,1,8,2,11,3,10,4,13,5,12,6,15,7,14},
											{ 0,10,7,13,14,4,9,3,15,5,8,2,1,11,6,12},
											{ 0,11,5,14,10,1,15,4,7,12,2,9,13,6,8,3},
											{ 0,12,11,7,5,9,14,2,10,6,1,13,15,3,4,8},
											{ 0,13,9,4,1,12,8,5,2,15,11,6,3,14,10,7},
											{ 0,14,15,1,13,3,2,12,9,7,6,8,4,10,11,5},
											{ 0,15,13,2,9,6,4,11,1,14,12,3,8,7,5,10}};

		const unsigned char inv[16] = { 0,1,9,14,13,11,7,6,15,2,12,5,10,4,3,8};
	#elif Q==32
		const unsigned char add[32][32] = {	{0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31},
											{1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14,17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30},
											{2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13,18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29},
											{3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12,19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28},
											{4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11,20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27},
											{5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10,21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26},
											{6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9,22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25},
											{7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8,23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24},
											{8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7,24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23},
											{9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6,25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22},
											{10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5,26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21},
											{11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4,27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20},
											{12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3,28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19},
											{13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2,29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18},
											{14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1,30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17},
											{15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0,31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16},
											{16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15},
											{17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30,1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14},
											{18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29,2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13},
											{19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28,3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12},
											{20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27,4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11},
											{21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26,5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10},
											{22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25,6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9},
											{23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24,7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8},
											{24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23,8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7},
											{25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22,9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6},
											{26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21,10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5},
											{27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20,11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4},
											{28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19,12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3},
											{29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18,13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2},
											{30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17,14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1},
											{31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0}};


		const unsigned char mult[32][32] = {{0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
											{0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31},
											{0,2,4,6,8,10,12,14,16,18,20,22,24,26,28,30,5,7,1,3,13,15,9,11,21,23,17,19,29,31,25,27},
											{0,3,6,5,12,15,10,9,24,27,30,29,20,23,18,17,21,22,19,16,25,26,31,28,13,14,11,8,1,2,7,4},
											{0,4,8,12,16,20,24,28,5,1,13,9,21,17,29,25,10,14,2,6,26,30,18,22,15,11,7,3,31,27,23,19},
											{0,5,10,15,20,17,30,27,13,8,7,2,25,28,19,22,26,31,16,21,14,11,4,1,23,18,29,24,3,6,9,12},
											{0,6,12,10,24,30,20,18,21,19,25,31,13,11,1,7,15,9,3,5,23,17,27,29,26,28,22,16,2,4,14,8},
											{0,7,14,9,28,27,18,21,29,26,19,20,1,6,15,8,31,24,17,22,3,4,13,10,2,5,12,11,30,25,16,23},
											{0,8,16,24,5,13,21,29,10,2,26,18,15,7,31,23,20,28,4,12,17,25,1,9,30,22,14,6,27,19,11,3},
											{0,9,18,27,1,8,19,26,2,11,16,25,3,10,17,24,4,13,22,31,5,12,23,30,6,15,20,29,7,14,21,28},
											{0,10,20,30,13,7,25,19,26,16,14,4,23,29,3,9,17,27,5,15,28,22,8,2,11,1,31,21,6,12,18,24},
											{0,11,22,29,9,2,31,20,18,25,4,15,27,16,13,6,1,10,23,28,8,3,30,21,19,24,5,14,26,17,12,7},
											{0,12,24,20,21,25,13,1,15,3,23,27,26,22,2,14,30,18,6,10,11,7,19,31,17,29,9,5,4,8,28,16},
											{0,13,26,23,17,28,11,6,7,10,29,16,22,27,12,1,14,3,20,25,31,18,5,8,9,4,19,30,24,21,2,15},
											{0,14,28,18,29,19,1,15,31,17,3,13,2,12,30,16,27,21,7,9,6,8,26,20,4,10,24,22,25,23,5,11},
											{0,15,30,17,25,22,7,8,23,24,9,6,14,1,16,31,11,4,21,26,18,29,12,3,28,19,2,13,5,10,27,20},
											{0,16,5,21,10,26,15,31,20,4,17,1,30,14,27,11,13,29,8,24,7,23,2,18,25,9,28,12,19,3,22,6},
											{0,17,7,22,14,31,9,24,28,13,27,10,18,3,21,4,29,12,26,11,19,2,20,5,1,16,6,23,15,30,8,25},
											{0,18,1,19,2,16,3,17,4,22,5,23,6,20,7,21,8,26,9,27,10,24,11,25,12,30,13,31,14,28,15,29},
											{0,19,3,16,6,21,5,22,12,31,15,28,10,25,9,26,24,11,27,8,30,13,29,14,20,7,23,4,18,1,17,2},
											{0,20,13,25,26,14,23,3,17,5,28,8,11,31,6,18,7,19,10,30,29,9,16,4,22,2,27,15,12,24,1,21},
											{0,21,15,26,30,11,17,4,25,12,22,3,7,18,8,29,23,2,24,13,9,28,6,19,14,27,1,20,16,5,31,10},
											{0,22,9,31,18,4,27,13,1,23,8,30,19,5,26,12,2,20,11,29,16,6,25,15,3,21,10,28,17,7,24,14},
											{0,23,11,28,22,1,29,10,9,30,2,21,31,8,20,3,18,5,25,14,4,19,15,24,27,12,16,7,13,26,6,17},
											{0,24,21,13,15,23,26,2,30,6,11,19,17,9,4,28,25,1,12,20,22,14,3,27,7,31,18,10,8,16,29,5},
											{0,25,23,14,11,18,28,5,22,15,1,24,29,4,10,19,9,16,30,7,2,27,21,12,31,6,8,17,20,13,3,26},
											{0,26,17,11,7,29,22,12,14,20,31,5,9,19,24,2,28,6,13,23,27,1,10,16,18,8,3,25,21,15,4,30},
											{0,27,19,8,3,24,16,11,6,29,21,14,5,30,22,13,12,23,31,4,15,20,28,7,10,17,25,2,9,18,26,1},
											{0,28,29,1,31,3,2,30,27,7,6,26,4,24,25,5,19,15,14,18,12,16,17,13,8,20,21,9,23,11,10,22},
											{0,29,31,2,27,6,4,25,19,14,12,17,8,21,23,10,3,30,28,1,24,5,7,26,16,13,15,18,11,22,20,9},
											{0,30,25,7,23,9,14,16,11,21,18,12,28,2,5,27,22,8,15,17,1,31,24,6,29,3,4,26,10,20,19,13},
											{0,31,27,4,19,12,8,23,3,28,24,7,16,15,11,20,6,25,29,2,21,10,14,17,5,26,30,1,22,9,13,18}};

		const unsigned char inv[32] = { 0,1,18,28,9,23,14,12,22,4,25,16,7,15,6,13,11,24,2,29,30,26,8,5,17,10,21,31,3,19,20,27};

	#endif
	


	for (int row = 0; row < M; row++){
		int col = col_row[row][0];
		for (int index = 0; index < row_weight[row]; index++){
			col = col_row[row][index];
			for (int a = 0; a < Q; a++) {
				ALPHAmn_[(Q*N*row)+(Q*col)+a] = GAMMAn_[(col*Q)+a];

			}
		}
	}


	/* for (int col = 0; col < N; col++){
		for (int a = 0; a < Q; a++) {
			printf("Gamma[%d][%d]=%d\n",col,a,GAMMAn_[(col*Q)+a]);

		}
	} */

	/* for (int row = 0; row < M; row++){
		for (int col = 0; col < N; col++){
			for (int a = 0; a < Q; a++) {
				printf("alpha[%d][%d][%d]=%d\n",row,col,a,ALPHAmn_[(Q*N*row)+(Q*col)+a]);

			}
		}
	} */
	
	#if Q==32

		unsigned char *h_nb=NULL;

		h_nb=(unsigned char *)malloc(sizeof(unsigned char)*M*N);
		if(h_nb == NULL){
			printf("Failed to allocate host h_nb\n" );
			exit(EXIT_FAILURE);
		}

		for (int row = 0; row < M; row++){
			for (int index = 0; index < N; index++){
				h_nb[(row*N)+index] = H_nb[row][index];
				//printf("H_nb[%d][%d]=%d\n", row, index, h_nb[row][index]);
			}
		}

	#else
		unsigned char h_nb[M][N];

		for (int row = 0; row < M; row++){
			for (int index = 0; index < N; index++){
				h_nb[row][index] = H_nb[row][index];
				//printf("H_nb[%d][%d]=%d\n", row, index, h_nb[row][index]);
			}
		}

	#endif

	free(H_nb);

	unsigned char w_row=row_weight[0];
	unsigned char w_col=col_weight[0];


	

	hipError_t err=hipSuccess; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	//======================================================================================================================================================================
	//kernel dimensions

		#if Q==4
			dim3 FB_threadsPerBlock(M,1,Q);
			dim3 FB_numBlocks(3,1,1);

			dim3 CN_threadsPerBlock(M,w_row,Q);
			dim3 CN_numBlocks(1,1,1);

			dim3 VN_threadsPerBlock(w_col,N,Q);
			dim3 VN_numBlocks(1,1,1);
		#elif Q==8
			dim3 FB_threadsPerBlock(M,1,Q);
			dim3 FB_numBlocks(3,1,1);

			dim3 CN_threadsPerBlock(M,w_row,Q);
			dim3 CN_numBlocks(1,1,1);

			dim3 VN_threadsPerBlock(w_col,N,Q);
			dim3 VN_numBlocks(1,1,1);
		#elif Q==16
			dim3 FB_threadsPerBlock(64,1,Q);
			dim3 FB_numBlocks(3,1,1);

			dim3 CN_threadsPerBlock(10,w_row,Q);
			dim3 CN_numBlocks(15,1,1);

			dim3 VN_threadsPerBlock(w_col,21,Q);
			dim3 VN_numBlocks(1,15,1);
		#elif Q==32
			dim3 FB_threadsPerBlock(32,1,Q);
			dim3 FB_numBlocks(10,1,1);

			dim3 CN_threadsPerBlock(5,w_row,Q);
			dim3 CN_numBlocks(62,1,1);

			dim3 VN_threadsPerBlock(w_col,10,Q);
			dim3 VN_numBlocks(1,62,1);
		#endif
  
	//======================================================================================================================================================================
	//size of variables
		size_t size_dimension=sizeof(unsigned short);
		size_t size_col_row=sizeof(unsigned short)*M*w_row;
		size_t size_row_col=sizeof(unsigned short)*w_col*N;
		size_t size_gamma=sizeof(unsigned char)*Q*N;
		size_t size_alpha=sizeof(unsigned char)*M*N*Q;
		size_t size_FB=sizeof(unsigned char)*M*w_row*Q;
		size_t size_GF=sizeof(unsigned char)*Q*Q;
		size_t size_GF_inv=sizeof(unsigned char)*Q;
		size_t size_h_nb=sizeof(unsigned char)*M*N;
		size_t size_weight=sizeof(unsigned char);
		size_t size_alpha_t=sizeof(unsigned char)*N*w_col*Q;
	
	//======================================================================================================================================================================
    //variables declaration
		unsigned char *d_gamma=NULL;
		unsigned char *d_alpha=NULL;
		unsigned char *d_F=NULL;
		unsigned char *d_B=NULL;
		unsigned char *d_beta=NULL;
		unsigned char *d_alpha_t=NULL;
		unsigned char *d_alpha_t2=NULL;
		#if Q==32
			unsigned char *d_h_nb=NULL;
		#endif
				
	//======================================================================================================================================================================
	//allocate host memory

		/* h_F=(unsigned char *)malloc(size_FB);
		if(h_F == NULL){
			printf("Failed to allocate host F\n" );
			exit(EXIT_FAILURE);
		}

		h_B=(unsigned char *)malloc(size_FB);
		if(h_B == NULL){
			printf("Failed to allocate host B\n" );
			exit(EXIT_FAILURE);
		} */

		

	//======================================================================================================================================================================
    //allocate device memory
		err=hipMalloc((void **)&d_gamma, size_gamma);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device gamma (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_alpha, size_alpha);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device alpha (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_F, size_FB);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device F (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_B, size_FB);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device B (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_beta, size_alpha);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device beta (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_alpha_t, size_alpha_t);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device alpha_t (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_alpha_t2, size_alpha_t);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device alpha_t2 (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		#if Q==32
			err=hipMalloc((void **)&d_h_nb, size_h_nb);
			if(err!=hipSuccess){
				fprintf(stderr, "Failed to allocate device h_nb (error code %d)!\n", hipGetLastError());
				exit(EXIT_FAILURE);
			}
		#endif

	//======================================================================================================================================================================
    //device memory initialization
		err=hipMemset(d_gamma, 0, size_gamma);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device gamma (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_alpha, 0, size_alpha);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device alpha (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_F, 0, size_FB);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device F (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_B, 0, size_FB);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device B (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_beta, 0, size_alpha);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device beta (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

				
	//======================================================================================================================================================================
	//copy data to constant


		err=hipMemcpyToSymbol(HIP_SYMBOL(c_col_row), &h_col_row, size_col_row);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy col_row from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_row_col), &h_row_col, size_row_col);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy row_col from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_M), &M, size_dimension,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy M from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_N), &N, size_dimension,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy N from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_add), &add, size_GF,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy add from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_mult), &mult, size_GF,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy mult from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		#if Q!=32
			err=hipMemcpyToSymbol(HIP_SYMBOL(c_h_nb), &h_nb, size_h_nb,0,hipMemcpyHostToDevice);
			if(err!=hipSuccess){
				fprintf(stderr, "Failed to copy H from host to constant (error code %d)!\n", hipGetLastError());
				exit(EXIT_FAILURE);
			}
		#endif

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_w_col), &w_col, size_weight,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy w_col from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_w_row), &w_row, size_weight,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy w_row from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_inv), &inv, size_GF_inv,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy w_row from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

	//======================================================================================================================================================================
	//copy data to device  
	
		err=hipMemcpy(d_gamma, GAMMAn_, size_gamma, hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy gamma from host to device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(d_alpha, ALPHAmn_, size_alpha, hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy alpha from host to device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		#if Q==32
			err=hipMemcpy(d_h_nb, h_nb, size_h_nb, hipMemcpyHostToDevice);
			if(err!=hipSuccess){
				fprintf(stderr, "Failed to copy alpha from host to device (error code %d)!\n", hipGetLastError());
				exit(EXIT_FAILURE);
			}
		#endif

	//======================================================================================================================================================================
	//execute the kernel
		hipEventRecord(start);
		for (int iter=0; iter < MAX_ITERATION; iter++) {
			#if Q==32
				GPU_FB_metrics<<<FB_numBlocks, FB_threadsPerBlock>>>( d_alpha, d_F, d_B, iter, d_h_nb);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
				}

				GPU_CN<<<CN_numBlocks, CN_threadsPerBlock>>>( d_beta, d_F, d_B, iter, d_h_nb);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
				} 

				GPU_VN<<<VN_numBlocks, VN_threadsPerBlock>>>( d_beta, d_alpha, d_alpha_t, d_alpha_t2, d_gamma, iter);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
				} 

			#else
				GPU_FB_metrics<<<FB_numBlocks, FB_threadsPerBlock>>>( d_alpha, d_F, d_B, iter);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
				}

				GPU_CN<<<CN_numBlocks, CN_threadsPerBlock>>>( d_beta, d_F, d_B, iter);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
				} 

				GPU_VN<<<VN_numBlocks, VN_threadsPerBlock>>>( d_beta, d_alpha, d_alpha_t, d_alpha_t2, d_gamma, iter);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
			} 
			#endif 
		}
		hipEventRecord(stop);


		

	//======================================================================================================================================================================
	//copy the data from device to host

	 	err=hipMemcpy(h_F, d_F, size_FB, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the F from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(h_B, d_B, size_FB, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the F from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(h_beta, d_beta, size_alpha, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the betta from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(h_alpha, d_alpha, size_alpha, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the betta from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		hipEventSynchronize(stop);

	//======================================================================================================================================================================
	//free the device memory
		err=hipFree(d_gamma);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the gamma from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_alpha);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the alpha from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_F);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the F from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_B);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the B from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_beta);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the beta from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_alpha_t);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the alpha_t from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_alpha_t2);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the alpha_t2 from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		#if Q==32
			err=hipFree(d_h_nb);
			if(err!=hipSuccess){
				fprintf(stderr, "Failed to free the H-nb from the device (error code %d)!\n", hipGetLastError());
				exit(EXIT_FAILURE);
			}
		#endif

	//======================================================================================================================================================================
	//save data
		
		

		/* for (int row = 0; row < M; row++) {
			for (int index_c = 0; index_c < w_row; index_c++) {	//2. update BETA values from B and F values
				for (int a = 0; a < Q; a++) {
					//printf("F[%d][%d][%d]=%d\n",row, a,index_c,h_F[(row*Q*w_row)+(a*w_row)+index_c]);
					printf("B[%d][%d][%d]=%d\n",row, a,w_row - index_c -1,h_B[(row*Q*w_row)+(a*w_row)+w_row - index_c -1]);
				}
			}
		} */

	//======================================================================================================================================================================
	//free the host memory
		
	hipEventElapsedTime(&milliseconds, start, stop);

	#if Q==32
		free(h_nb);
	#endif

			
	return 0;
}
