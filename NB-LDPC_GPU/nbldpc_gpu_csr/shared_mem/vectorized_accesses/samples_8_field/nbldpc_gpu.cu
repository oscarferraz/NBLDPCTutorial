#include "hip/hip_runtime.h"

#include "stdio.h"
#include "stdlib.h"
#include <math.h>
#include <time.h>
#include <string.h>
//#include <ap_fixed.h>
#include "nbldpc.h"
#include <hip/hip_runtime.h>      // CUDA Runtime Functions
#include <hip/hip_runtime_api.h>

extern "C" {
    #include "nbldpc_gpu.h"
}


#ifdef VS
#define _CRT_SECURE_NO_DEPRECATE
#endif


__constant__ unsigned char c_add[Q][Q];
__constant__ unsigned char c_mult[Q][Q];
__constant__ unsigned char c_inv[Q];


__constant__ unsigned short	c_row_ptr[257];
__constant__ unsigned char	c_val[768];
__constant__ unsigned short	c_ptr_to_val[768];
__constant__ unsigned short	c_col_ptr[385];
__constant__ unsigned short	c_row_ind[768];
__constant__ unsigned short	c_col_ind[768];





__constant__ unsigned short c_M;
__constant__ unsigned short c_N;
__constant__ unsigned char c_w_row;
__constant__ unsigned char c_w_col;

extern unsigned char *h_F;
extern unsigned char *h_B;



    //===================================
	// CUDA kernel
	//===================================
	__global__ void GPU_FB_metrics( uchar8 * d_alpha_8,  uchar8 * d_F_8,  uchar8 * d_B_8, int iter){
		
		unsigned short x=threadIdx.x+blockIdx.x*blockDim.x;
		uchar8 min_F[1];
		uchar8 max_F;
		uchar8 min_B[1];
		uchar8 max_B;
		uchar8 a_F;
		uchar8 a_B;

		#if Q==8
		__shared__ uchar8 s_alpha[128*3];
		__shared__ uchar8 s_F[128*3];
		__shared__ uchar8 s_B[128*3];
		#elif Q==16
		__shared__ uchar16 s_alpha[8*3*16];
		__shared__ uchar16 s_F[8*16*3];
		__shared__ uchar16 s_B[8*16*3];
		#elif Q==32
		__shared__ uchar16 s_alpha[8*3*32];
		__shared__ uchar16 s_F[8*32*3];
		__shared__ uchar16 s_B[8*32*3];
		#endif

		//if(x<c_M){

			for(unsigned char index=0; index<c_w_row; index++){
				reinterpret_cast<uchar8*>(s_alpha)[(threadIdx.x*c_w_row)+index]=reinterpret_cast<uchar8*>(d_alpha_8)[(x*c_w_row)+index];
				/* if(x==0){
					printf("s_alpha[%d][%d][0]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0, d_alpha_8[(x*c_w_row)+index].s0);  
					printf("s_alpha[%d][%d][1]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s1, d_alpha_8[(x*c_w_row)+index].s1);  
					printf("s_alpha[%d][%d][2]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s2, d_alpha_8[(x*c_w_row)+index].s2);  
					printf("s_alpha[%d][%d][3]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s3, d_alpha_8[(x*c_w_row)+index].s3);
					printf("s_alpha[%d][%d][4]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s4, d_alpha_8[(x*c_w_row)+index].s4);  
					printf("s_alpha[%d][%d][5]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s5, d_alpha_8[(x*c_w_row)+index].s5);  
					printf("s_alpha[%d][%d][6]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s6, d_alpha_8[(x*c_w_row)+index].s6);  
					printf("s_alpha[%d][%d][7]=%d col=%d\n", threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s7, d_alpha_8[(x*c_w_row)+index].s7);  
				}  */
			}
			__syncthreads();

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==0)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==1)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==2)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==3)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==4)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==5)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==6)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][0]==7)
				s_F[threadIdx.x*c_w_row].s0=s_alpha[threadIdx.x*c_w_row].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==0)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==1)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==2)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==3)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==4)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==5)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==6)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][1]==7)
				s_F[threadIdx.x*c_w_row].s1=s_alpha[threadIdx.x*c_w_row].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==0)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==1)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==2)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==3)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==4)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==5)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==6)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][2]==7)
				s_F[threadIdx.x*c_w_row].s2=s_alpha[threadIdx.x*c_w_row].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==0)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==1)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==2)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==3)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==4)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==5)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==6)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][3]==7)
				s_F[threadIdx.x*c_w_row].s3=s_alpha[threadIdx.x*c_w_row].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==0)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==1)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==2)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==3)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==4)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==5)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==6)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][4]==7)
				s_F[threadIdx.x*c_w_row].s4=s_alpha[threadIdx.x*c_w_row].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==0)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==1)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==2)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==3)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==4)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==5)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==6)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][5]==7)
				s_F[threadIdx.x*c_w_row].s5=s_alpha[threadIdx.x*c_w_row].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==0)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==1)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==2)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==3)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==4)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==5)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==6)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][6]==7)
				s_F[threadIdx.x*c_w_row].s6=s_alpha[threadIdx.x*c_w_row].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==0)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==1)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==2)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==3)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==4)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==5)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==6)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x]]]][7]==7)
				s_F[threadIdx.x*c_w_row].s7=s_alpha[threadIdx.x*c_w_row].s7;

			/* if(x==0){
				printf("s_alpha[%d][0]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s0, c_mult[c_inv[c_val[c_row_ptr[x]]]][0]);  
				printf("s_alpha[%d][1]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s1, c_mult[c_inv[c_val[c_row_ptr[x]]]][1]);  
				printf("s_alpha[%d][2]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s2, c_mult[c_inv[c_val[c_row_ptr[x]]]][2]);  
				printf("s_alpha[%d][3]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s3, c_mult[c_inv[c_val[c_row_ptr[x]]]][3]); 
				printf("s_alpha[%d][4]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s4, c_mult[c_inv[c_val[c_row_ptr[x]]]][4]);  
				printf("s_alpha[%d][5]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s5, c_mult[c_inv[c_val[c_row_ptr[x]]]][5]);  
				printf("s_alpha[%d][6]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s6, c_mult[c_inv[c_val[c_row_ptr[x]]]][6]);  
				printf("s_alpha[%d][7]=%d mult=%d\n", threadIdx.x, s_F[threadIdx.x*c_w_row].s7, c_mult[c_inv[c_val[c_row_ptr[x]]]][7]);   
			} */


			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][0]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;


			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][1]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][2]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][3]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][4]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s4=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][5]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s5=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][6]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s6=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;

			if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==0)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s0;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==1)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s1;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==2)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s2;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==3)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s3;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==4)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s4;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==5)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s5;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==6)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s6;
			else if(c_mult[c_inv[c_val[c_row_ptr[x+1]-1]]][7]==7)
				s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s7=s_alpha[(threadIdx.x*c_w_row)+(c_w_row-1)].s7;
			/* if(x==6){
				printf("s_B[%d][0][%d]=%d \n", (threadIdx.x*c_w_row), (c_w_row-1), s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s0);  
				printf("s_B[%d][1][%d]=%d \n", (threadIdx.x*c_w_row), (c_w_row-1), s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s1);  
				printf("s_B[%d][2][%d]=%d \n", (threadIdx.x*c_w_row), (c_w_row-1), s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s2);  
				printf("s_B[%d][3][%d]=%d \n", (threadIdx.x*c_w_row), (c_w_row-1), s_B[(threadIdx.x*c_w_row)+(c_w_row-1)].s3);  
			} */

			reinterpret_cast<uchar8*>(d_F_8)[(x*c_w_row)]=reinterpret_cast<uchar8*>(s_F)[(threadIdx.x*c_w_row)];                   
			/* if(x==0){
				printf("s_F[%d][0]=%d \n", x, d_F_8[(x*c_w_row)].s0);  
				printf("s_F[%d][1]=%d \n", x, d_F_8[(x*c_w_row)].s1);  
				printf("s_F[%d][2]=%d \n", x, d_F_8[(x*c_w_row)].s2);  
				printf("s_F[%d][3]=%d \n", x, d_F_8[(x*c_w_row)].s3);  
				printf("s_F[%d][4]=%d \n", x, d_F_8[(x*c_w_row)].s4);  
				printf("s_F[%d][5]=%d \n", x, d_F_8[(x*c_w_row)].s5);  
				printf("s_F[%d][6]=%d \n", x, d_F_8[(x*c_w_row)].s6);  
				printf("s_F[%d][7]=%d \n", x, d_F_8[(x*c_w_row)].s7);  
			} */

			reinterpret_cast<uchar8*>(d_B_8)[(x*c_w_row)+(c_w_row-1)]=reinterpret_cast<uchar8*>(s_B)[(threadIdx.x*c_w_row)+(c_w_row-1)];                   
			__syncthreads();

			for(unsigned char index=1; index<c_w_row; index++){
				
				min_F[0].s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
				min_F[0].s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
				min_F[0].s2 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
				min_F[0].s3 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
				min_F[0].s4 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
				min_F[0].s5 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
				min_F[0].s6 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
				min_F[0].s7 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
				/* if(x==0){
					printf("min_F=%d, s_F[%d][0][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s0, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s0,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );  
					printf("min_F=%d, s_F[%d][1][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s1, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s1,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );  
					printf("min_F=%d, s_F[%d][2][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s2, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s2,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );  
					printf("min_F=%d, s_F[%d][3][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s3, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s3,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );
					printf("min_F=%d, s_F[%d][4][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s4, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s4,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );  
					printf("min_F=%d, s_F[%d][5][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s5, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s5,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );  
					printf("min_F=%d, s_F[%d][6][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s6, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s6,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );  
					printf("min_F=%d, s_F[%d][7][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_F[0].s7, threadIdx.x, index-1,s_F[(threadIdx.x*c_w_row)+index-1].s7,threadIdx.x, index, s_alpha[(threadIdx.x*c_w_row)+index].s0 );  
				} */

				min_B[0].s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
				min_B[0].s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
				min_B[0].s2 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
				min_B[0].s3 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
				min_B[0].s4 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
				min_B[0].s5 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
				min_B[0].s6 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
				min_B[0].s7 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
				/* if(x==0){
					printf("min_B=%d, s_B[%d][0][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_B[0].s0, threadIdx.x, c_w_row-index, s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0, threadIdx.x, c_w_row-index-1, s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0 );  
					printf("min_B=%d, s_B[%d][1][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_B[0].s1, threadIdx.x, c_w_row-index, s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1, threadIdx.x, c_w_row-index-1, s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0 );  
					printf("min_B=%d, s_B[%d][2][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_B[0].s2, threadIdx.x, c_w_row-index, s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2, threadIdx.x, c_w_row-index-1, s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0 );  
					printf("min_B=%d, s_B[%d][3][%d]=%d, s_alpha[%d][%d][0]=%d\n", min_B[0].s3, threadIdx.x, c_w_row-index, s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3, threadIdx.x, c_w_row-index-1, s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0 );  
				} */

				for(unsigned char b=1; b<Q; b++){
					a_F.s0 = c_add[0][c_mult[c_val[c_row_ptr[x]+index]][b]];
					a_F.s1 = c_add[1][c_mult[c_val[c_row_ptr[x]+index]][b]];
					a_F.s2 = c_add[2][c_mult[c_val[c_row_ptr[x]+index]][b]];
					a_F.s3 = c_add[3][c_mult[c_val[c_row_ptr[x]+index]][b]];
					a_F.s4 = c_add[4][c_mult[c_val[c_row_ptr[x]+index]][b]];
					a_F.s5 = c_add[5][c_mult[c_val[c_row_ptr[x]+index]][b]];
					a_F.s6 = c_add[6][c_mult[c_val[c_row_ptr[x]+index]][b]];
					a_F.s7 = c_add[7][c_mult[c_val[c_row_ptr[x]+index]][b]];
					/* if(x==0){
						printf("a_F=%d, z=0, index=%d, b=%d\n", a_F.s0, index, b );  
						printf("a_F=%d, z=1, index=%d, b=%d\n", a_F.s1, index, b );  
						printf("a_F=%d, z=2, index=%d, b=%d\n", a_F.s2, index, b );  
						printf("a_F=%d, z=3, index=%d, b=%d\n", a_F.s3, index, b );  
						printf("a_F=%d, z=4, index=%d, b=%d\n", a_F.s4, index, b );  
						printf("a_F=%d, z=5, index=%d, b=%d\n", a_F.s5, index, b );  
						printf("a_F=%d, z=6, index=%d, b=%d\n", a_F.s6, index, b );  
						printf("a_F=%d, z=7, index=%d, b=%d\n", a_F.s7, index, b );  
					} */

					a_B.s0 = c_add[0][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					a_B.s1 = c_add[1][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					a_B.s2 = c_add[2][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					a_B.s3 = c_add[3][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					a_B.s4 = c_add[4][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					a_B.s5 = c_add[5][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					a_B.s6 = c_add[6][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					a_B.s7 = c_add[7][c_mult[c_val[c_row_ptr[x]+c_row_ptr[x+1]-(c_row_ptr[x]+index)-1]][b]];
					/* if(x==0){
						printf("a_B=%d, z=0, index=%d, b=%d\n", a_B.x, index, b );  
						printf("a_B=%d, z=1, index=%d, b=%d\n", a_B.y, index, b );  
						printf("a_B=%d, z=2, index=%d, b=%d\n", a_B.z, index, b );  
						printf("a_B=%d, z=3, index=%d, b=%d\n", a_B.w, index, b );  
					} */
					
					// FORWARD
					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S0
						if(a_F.s0==0){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s0==1){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s0==2){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s0==3){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s0==4){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s0==5){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s0==6){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s0==7){
							if(b==0)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s0 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S1
						if(a_F.s1==0){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s1==1){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s1==2){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s1==3){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s1==4){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s1==5){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s1==6){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s1==7){
							if(b==0)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s1 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S2
						if(a_F.s2==0){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s2==1){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s2==2){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s2==3){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s2==4){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s2==5){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s2==6){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s2==7){
							if(b==0)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s2= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s2 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S3
						if(a_F.s3==0){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s3==1){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s3==2){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s3==3){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s3==4){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s3==5){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s3==6){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s3==7){
							if(b==0)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s3= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s3 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S4
						if(a_F.s4==0){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s4==1){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s4==2){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s4==3){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s4==4){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s4==5){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s4==6){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s4==7){
							if(b==0)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s4= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S5
						if(a_F.s5==0){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s5==1){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s5==2){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s5==3){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s5==4){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s5==5){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s5==6){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s5==7){
							if(b==0)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s5= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s5 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S6
						if(a_F.s6==0){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s6==1){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s6==2){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s6==3){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s6==4){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s6==5){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s6==6){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s6==7){
							if(b==0)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s6= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S7
						if(a_F.s7==0){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else  if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s0;
							else if(b==7)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s0 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s0;
						}
						else if(a_F.s7==1){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s1;
							else if(b==7)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s1 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s1;
						}
						else if(a_F.s7==2){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s2;
							else if(b==7)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s2 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s2;
						}
						else if(a_F.s7==3){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s3;
							else if(b==7)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s3 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s3;
						}

						else if(a_F.s7==4){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else  if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s4;
							else if(b==7)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s4 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s4;
						}
						else if(a_F.s7==5){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s5;
							else if(b==7)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s5 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s5;
						}
						else if(a_F.s7==6){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s6;
							else if(b==7)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s6 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s6;
						}
						else if(a_F.s7==7){
							if(b==0)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s0) ? s_alpha[(threadIdx.x*c_w_row)+index].s0: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==1)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s1) ? s_alpha[(threadIdx.x*c_w_row)+index].s1: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==2)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s2) ? s_alpha[(threadIdx.x*c_w_row)+index].s2: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==3)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s3) ? s_alpha[(threadIdx.x*c_w_row)+index].s3: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==4)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s4) ? s_alpha[(threadIdx.x*c_w_row)+index].s4: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==5)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s5) ? s_alpha[(threadIdx.x*c_w_row)+index].s5: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==6)
								max_F.s7= (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s6) ? s_alpha[(threadIdx.x*c_w_row)+index].s6: s_F[(threadIdx.x*c_w_row)+index-1].s7;
							else if(b==7)
								max_F.s7 = (s_F[(threadIdx.x*c_w_row)+index-1].s7 < s_alpha[(threadIdx.x*c_w_row)+index].s7) ? s_alpha[(threadIdx.x*c_w_row)+index].s7: s_F[(threadIdx.x*c_w_row)+index-1].s7;
						}
					/* if(x==0){
						printf("max_F=%d, z=0, index=%d, b=%d\n", max_F.s0, index, b );  
						printf("max_F=%d, z=1, index=%d, b=%d\n", max_F.s1, index, b );  
						printf("max_F=%d, z=2, index=%d, b=%d\n", max_F.s2, index, b );  
						printf("max_F=%d, z=3, index=%d, b=%d\n", max_F.s3, index, b );  
						printf("max_F=%d, z=4, index=%d, b=%d\n", max_F.s4, index, b );  
						printf("max_F=%d, z=5, index=%d, b=%d\n", max_F.s5, index, b );  
						printf("max_F=%d, z=6, index=%d, b=%d\n", max_F.s6, index, b );  
						printf("max_F=%d, z=7, index=%d, b=%d\n", max_F.s7, index, b );  
					} */

					// BACKWARD
					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S0
						if(a_B.s0==0){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s0==1){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s0==2){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s0==3){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s0==4){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s0==5){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s0==6){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s0==7){
							if(b==0)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s0 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S1
						if(a_B.s1==0){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s1==1){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s1==2){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s1==3){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s1==4){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s1==5){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s1==6){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s1==7){
							if(b==0)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s1 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S2
						if(a_B.s2==0){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s2==1){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s2==2){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s2==3){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s2==4){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s2==5){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s2==6){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s2==7){
							if(b==0)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s2= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s2 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S3
						if(a_B.s3==0){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s3==1){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s3==2){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s3==3){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s3==4){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s3==5){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s3==6){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s3==7){
							if(b==0)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s3= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s3 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S4
						if(a_B.s4==0){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s4==1){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s4==2){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s4==3){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s4==4){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s4==5){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s4==6){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s4==7){
							if(b==0)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s4= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s4 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S5
						if(a_B.s5==0){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s5==1){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s5==2){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s5==3){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s5==4){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s5==5){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s5==6){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s5==7){
							if(b==0)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s5= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s5 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S6
						if(a_B.s6==0){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s6==1){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s6==2){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s6==3){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s6==4){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s6==5){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s6==6){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s6==7){
							if(b==0)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s6= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s6 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S7
						if(a_B.s7==0){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else  if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
							else if(b==7)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s0;
						}
						else if(a_B.s7==1){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
							else if(b==7)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s1;
						}
						else if(a_B.s7==2){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
							else if(b==7)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s2;
						}
						else if(a_B.s7==3){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
							else if(b==7)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s3;
						}

						else if(a_B.s7==4){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else  if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
							else if(b==7)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s4;
						}
						else if(a_B.s7==5){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
							else if(b==7)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s5;
						}
						else if(a_B.s7==6){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
							else if(b==7)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s6;
						}
						else if(a_B.s7==7){
							if(b==0)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s0: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==1)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s1: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==2)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s2: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==3)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s3: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==4)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s4: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==5)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s5: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==6)
								max_B.s7= (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s6: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
							else if(b==7)
								max_B.s7 = (s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7 < s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7) ? s_alpha[(threadIdx.x*c_w_row)+(c_w_row-index-1)].s7: s_B[(threadIdx.x*c_w_row)+c_w_row-index].s7;
						}

					/* if(x==0){
						printf("max_B=%d, z=0, index=%d, b=%d\n", max_B.x, index, b );  
						printf("max_B=%d, z=1, index=%d, b=%d\n", max_B.y, index, b );  
						printf("max_B=%d, z=2, index=%d, b=%d\n", max_B.z, index, b );  
						printf("max_B=%d, z=3, index=%d, b=%d\n", max_B.w, index, b );  
					} */

					min_F[0].s0 = (min_F[0].s0 < max_F.s0) ? min_F[0].s0: max_F.s0;
					min_F[0].s1 = (min_F[0].s1 < max_F.s1) ? min_F[0].s1: max_F.s1;
					min_F[0].s2 = (min_F[0].s2 < max_F.s2) ? min_F[0].s2: max_F.s2;
					min_F[0].s3 = (min_F[0].s3 < max_F.s3) ? min_F[0].s3: max_F.s3;
					min_F[0].s4 = (min_F[0].s4 < max_F.s4) ? min_F[0].s4: max_F.s4;
					min_F[0].s5 = (min_F[0].s5 < max_F.s5) ? min_F[0].s5: max_F.s5;
					min_F[0].s6 = (min_F[0].s6 < max_F.s6) ? min_F[0].s6: max_F.s6;
					min_F[0].s7 = (min_F[0].s7 < max_F.s7) ? min_F[0].s7: max_F.s7;
					/* if(x==0){
						printf("min_F[0]=%d, max_F=%d, z=0, index=%d, b=%d\n", min_F[0].s0, max_F.s0, index, b );  
						printf("min_F[0]=%d, max_F=%d, z=1, index=%d, b=%d\n", min_F[0].s1, max_F.s1, index, b );  
						printf("min_F[0]=%d, max_F=%d, z=2, index=%d, b=%d\n", min_F[0].s2, max_F.s2, index, b );  
						printf("min_F[0]=%d, max_F=%d, z=3, index=%d, b=%d\n", min_F[0].s3, max_F.s3, index, b ); 
						printf("min_F[0]=%d, max_F=%d, z=4, index=%d, b=%d\n", min_F[0].s4, max_F.s4, index, b );  
						printf("min_F[0]=%d, max_F=%d, z=5, index=%d, b=%d\n", min_F[0].s5, max_F.s5, index, b );  
						printf("min_F[0]=%d, max_F=%d, z=6, index=%d, b=%d\n", min_F[0].s6, max_F.s6, index, b );  
						printf("min_F[0]=%d, max_F=%d, z=7, index=%d, b=%d\n", min_F[0].s7, max_F.s7, index, b );  
					} */

					min_B[0].s0 = (min_B[0].s0 < max_B.s0) ? min_B[0].s0: max_B.s0;
					min_B[0].s1 = (min_B[0].s1 < max_B.s1) ? min_B[0].s1: max_B.s1;
					min_B[0].s2 = (min_B[0].s2 < max_B.s2) ? min_B[0].s2: max_B.s2;
					min_B[0].s3 = (min_B[0].s3 < max_B.s3) ? min_B[0].s3: max_B.s3;
					min_B[0].s4 = (min_B[0].s4 < max_B.s4) ? min_B[0].s4: max_B.s4;
					min_B[0].s5 = (min_B[0].s5 < max_B.s5) ? min_B[0].s5: max_B.s5;
					min_B[0].s6 = (min_B[0].s6 < max_B.s6) ? min_B[0].s6: max_B.s6;
					min_B[0].s7 = (min_B[0].s7 < max_B.s7) ? min_B[0].s7: max_B.s7;
					/* if(x==6){
						printf("min_F[0]=%d, z=0, index=%d, b=%d\n", min_B[0].s0, index, b );  
						printf("min_F[0]=%d, z=1, index=%d, b=%d\n", min_B[0].s1, index, b );  
						printf("min_F[0]=%d, z=2, index=%d, b=%d\n", min_B[0].s2, index, b );  
						printf("min_F[0]=%d, z=3, index=%d, b=%d\n", min_B[0].s3, index, b );  
					} */

				}
				__syncthreads();

				reinterpret_cast<uchar8*>(d_F_8)[(x*c_w_row)+index]=reinterpret_cast<uchar8*>(min_F)[0];                   
				s_F[(threadIdx.x*c_w_row)+index].s0 = min_F[0].s0;
				s_F[(threadIdx.x*c_w_row)+index].s1 = min_F[0].s1;
				s_F[(threadIdx.x*c_w_row)+index].s2 = min_F[0].s2;
				s_F[(threadIdx.x*c_w_row)+index].s3 = min_F[0].s3;
				s_F[(threadIdx.x*c_w_row)+index].s4 = min_F[0].s4;
				s_F[(threadIdx.x*c_w_row)+index].s5 = min_F[0].s5;
				s_F[(threadIdx.x*c_w_row)+index].s6 = min_F[0].s6;
				s_F[(threadIdx.x*c_w_row)+index].s7 = min_F[0].s7;
				/* if(x==0){
					printf("d_F_8[%d][0][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s0);  
					printf("d_F_8[%d][1][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s1);  
					printf("d_F_8[%d][2][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s2);  
					printf("d_F_8[%d][3][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s3); 
					printf("d_F_8[%d][4][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s4);  
					printf("d_F_8[%d][5][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s5);  
					printf("d_F_8[%d][6][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s6);  
					printf("d_F_8[%d][7][%d]=%d\n", x, index, d_F_8[(x*c_w_row)+index].s7);   
				} */

				reinterpret_cast<uchar8*>(d_B_8)[(x*c_w_row)+c_w_row - index - 1]=reinterpret_cast<uchar8*>(min_B)[0];                   
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s0 = min_B[0].s0;
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s1 = min_B[0].s1;
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s2 = min_B[0].s2;
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s3 = min_B[0].s3;
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s4 = min_B[0].s4;
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s5 = min_B[0].s5;
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s6 = min_B[0].s6;
				s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s7 = min_B[0].s7;
				/* if(x==6){
					printf("s_F[%d][0][%d]=%d\n", threadIdx.x, c_w_row - index - 1, s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s0);  
					printf("s_F[%d][1][%d]=%d\n", threadIdx.x, c_w_row - index - 1, s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s1);  
					printf("s_F[%d][2][%d]=%d\n", threadIdx.x, c_w_row - index - 1, s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s2);  
					printf("s_F[%d][3][%d]=%d\n", threadIdx.x, c_w_row - index - 1, s_B[(threadIdx.x*c_w_row)+c_w_row - index - 1].s3);  
				} */
			
			}  
		//}

		

	} 

	__global__ void GPU_CN( uchar8 * d_beta_8,  uchar8 * d_F_8,  uchar8 * d_B_8, int iter){
		unsigned int x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned int y=threadIdx.y+blockIdx.y*blockDim.y;		

		#if Q==8
		__shared__ uchar8 s_F[32*3];
		__shared__ uchar8 s_B[32*3];
		#elif Q==16
		__shared__ uchar16 s_F[4*16*3];
		__shared__ uchar16 s_B[4*16*3];
		#elif Q==32
		__shared__ uchar16 s_F[4*32*3];
		__shared__ uchar16 s_B[4*32*3];
		#endif

		//if(x<c_M){

			reinterpret_cast<uchar8*>(s_F)[(threadIdx.x*c_w_row)+y]=reinterpret_cast<uchar8*>(d_F_8)[(x*c_w_row)+y];
			reinterpret_cast<uchar8*>(s_B)[(threadIdx.x*c_w_row)+y]=reinterpret_cast<uchar8*>(d_B_8)[(x*c_w_row)+y];
			__syncthreads();
			/* if(x==1){
				printf("s_F[%d][0][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s0);  
				printf("s_F[%d][1][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s1);  
				printf("s_F[%d][2][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s2);  
				printf("s_F[%d][3][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s3);  
				printf("s_F[%d][4][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s4);  
				printf("s_F[%d][5][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s5);  
				printf("s_F[%d][6][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s6);  
				printf("s_F[%d][7][%d]=%d\n", threadIdx.x, y, s_F[(threadIdx.x*c_w_row)+y].s7); 
			}

			if(x==1){
				printf("s_B[%d][0][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s0);  
				printf("s_B[%d][1][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s1);  
				printf("s_B[%d][2][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s2);  
				printf("s_B[%d][3][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s3);  
				printf("s_B[%d][4][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s4);  
				printf("s_B[%d][5][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s5);  
				printf("s_B[%d][6][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s6);  
				printf("s_B[%d][7][%d]=%d\n", threadIdx.x, y, s_B[(threadIdx.x*c_w_row)+y].s7); 
			} */

			if(y==0){		
				if(c_mult[c_val[c_row_ptr[x]]][0]==0)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][0]==1)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][0]==2)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][0]==3)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][0]==4)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][0]==5)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][0]==6)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][0]==7)
					d_beta_8[(x*c_w_row)+y].s0=s_B[(threadIdx.x*c_w_row)+1].s7;
	
	
				if(c_mult[c_val[c_row_ptr[x]]][1]==0)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][1]==1)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][1]==2)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][1]==3)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][1]==4)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][1]==5)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][1]==6)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][1]==7)
					d_beta_8[(x*c_w_row)+y].s1=s_B[(threadIdx.x*c_w_row)+1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]]][2]==0)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][2]==1)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][2]==2)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][2]==3)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][2]==4)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][2]==5)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][2]==6)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][2]==7)
					d_beta_8[(x*c_w_row)+y].s2=s_B[(threadIdx.x*c_w_row)+1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]]][3]==0)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][3]==1)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][3]==2)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][3]==3)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][3]==4)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][3]==5)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][3]==6)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][3]==7)
					d_beta_8[(x*c_w_row)+y].s3=s_B[(threadIdx.x*c_w_row)+1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]]][4]==0)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][4]==1)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][4]==2)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][4]==3)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][4]==4)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][4]==5)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][4]==6)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][4]==7)
					d_beta_8[(x*c_w_row)+y].s4=s_B[(threadIdx.x*c_w_row)+1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]]][5]==0)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][5]==1)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][5]==2)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][5]==3)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][5]==4)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][5]==5)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][5]==6)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][5]==7)
					d_beta_8[(x*c_w_row)+y].s5=s_B[(threadIdx.x*c_w_row)+1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]]][6]==0)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][6]==1)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][6]==2)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][6]==3)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][6]==4)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][6]==5)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][6]==6)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][6]==7)
					d_beta_8[(x*c_w_row)+y].s6=s_B[(threadIdx.x*c_w_row)+1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]]][7]==0)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s0;
				else if(c_mult[c_val[c_row_ptr[x]]][7]==1)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s1;
				else if(c_mult[c_val[c_row_ptr[x]]][7]==2)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s2;
				else if(c_mult[c_val[c_row_ptr[x]]][7]==3)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s3;
				else if(c_mult[c_val[c_row_ptr[x]]][7]==4)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s4;
				else if(c_mult[c_val[c_row_ptr[x]]][7]==5)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s5;
				else if(c_mult[c_val[c_row_ptr[x]]][7]==6)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s6;
				else if(c_mult[c_val[c_row_ptr[x]]][7]==7)
					d_beta_8[(x*c_w_row)+y].s7=s_B[(threadIdx.x*c_w_row)+1].s7;
				/* if(x==3){
					printf("BETAmn_[%d][%d][0]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].x);  
					printf("BETAmn_[%d][%d][1]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].y);  
					printf("BETAmn_[%d][%d][2]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].z);  
					printf("BETAmn_[%d][%d][3]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].w);  
				} */

			}
			else if(y==c_w_row-1){
				if(c_mult[c_val[c_row_ptr[x]+y]][0]==0)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==1)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==2)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==3)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==4)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==5)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==6)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==7)
					d_beta_8[(x*c_w_row)+y].s0=s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
	
				if(c_mult[c_val[c_row_ptr[x]+y]][1]==0)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==1)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==2)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==3)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==4)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==5)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==6)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==7)
					d_beta_8[(x*c_w_row)+y].s1=s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][2]==0)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==1)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==2)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==3)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==4)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==5)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==6)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==7)
					d_beta_8[(x*c_w_row)+y].s2=s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][3]==0)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==1)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==2)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==3)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==4)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==5)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==6)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==7)
					d_beta_8[(x*c_w_row)+y].s3=s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][4]==0)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==1)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==2)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==3)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==4)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==5)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==6)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==7)
					d_beta_8[(x*c_w_row)+y].s4=s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][5]==0)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==1)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==2)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==3)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==4)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==5)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==6)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==7)
					d_beta_8[(x*c_w_row)+y].s5=s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][6]==0)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==1)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==2)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==3)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==4)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==5)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==6)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==7)
					d_beta_8[(x*c_w_row)+y].s6=s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][7]==0)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==1)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==2)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==3)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==4)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==5)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==6)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==7)
					d_beta_8[(x*c_w_row)+y].s7=s_F[(threadIdx.x*c_w_row)+y-1].s7;
				/* if(x==3){
					printf("BETAmn_[%d][%d][0]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].x);  
					printf("BETAmn_[%d][%d][1]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].y);  
					printf("BETAmn_[%d][%d][2]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].z);  
					printf("BETAmn_[%d][%d][3]=%d\n", x, y, d_beta_8[(x*c_w_row)+y].w);  
				} */

			}
			else{
				uchar8 min[1];
				uchar8 max;
				uchar8 a;

				if(c_mult[c_val[c_row_ptr[x]+y]][0]==0)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==1)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==2)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==3)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==4)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==5)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==6)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][0]==7)
					min[0].s0=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
	
				if(c_mult[c_val[c_row_ptr[x]+y]][1]==0)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==1)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==2)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==3)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==4)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==5)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==6)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][1]==7)
					min[0].s1=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][2]==0)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==1)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==2)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==3)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==4)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==5)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==6)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][2]==7)
					min[0].s2=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][3]==0)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==1)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==2)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==3)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==4)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==5)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==6)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][3]==7)
					min[0].s3=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][4]==0)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==1)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==2)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==3)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==4)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==5)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==6)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][4]==7)
					min[0].s4=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][5]==0)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==1)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==2)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==3)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==4)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==5)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==6)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][5]==7)
					min[0].s5=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][6]==0)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==1)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==2)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==3)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==4)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==5)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==6)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][6]==7)
					min[0].s6=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
	
				if(c_mult[c_val[c_row_ptr[x]+y]][7]==0)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s0;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==1)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s1;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==2)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s2;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==3)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s3;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==4)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s4;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==5)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s5;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==6)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s6;
				else if(c_mult[c_val[c_row_ptr[x]+y]][7]==7)
					min[0].s7=(s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0 : s_F[(threadIdx.x*c_w_row)+y-1].s7;
				/* if(x==1){
					printf("min[0]=%d, s_f=%d , s_b=%d\n", min[0].s0,s_F[(threadIdx.x*c_w_row)+y-1].s2, s_B[(threadIdx.x*c_w_row)+y+1].s2 );  
					printf("min[1]=%d\n", min[0].s1);  
					printf("min[2]=%d\n", min[0].s2);  
					printf("min[3]=%d\n", min[0].s3);  
					printf("min[4]=%d\n", min[0].s4);  
					printf("min[5]=%d\n", min[0].s5);  
					printf("min[6]=%d\n", min[0].s6);  
					printf("min[7]=%d\n", min[0].s7); 
				} */
				
				for(unsigned char b=1; b<Q; b++){
					if(c_mult[c_val[c_row_ptr[x]+y]][0]==0)
						a.s0=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][0]==1)
						a.s0=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][0]==2)
						a.s0=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][0]==3)
						a.s0=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][0]==4)
						a.s0=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][0]==5)
						a.s0=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][0]==6)
						a.s0=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][0]==7)
						a.s0=c_add[b][7];
		
		
					if(c_mult[c_val[c_row_ptr[x]+y]][1]==0)
						a.s1=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][1]==1)
						a.s1=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][1]==2)
						a.s1=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][1]==3)
						a.s1=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][1]==4)
						a.s1=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][1]==5)
						a.s1=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][1]==6)
						a.s1=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][1]==7)
						a.s1=c_add[b][7];
		
					if(c_mult[c_val[c_row_ptr[x]+y]][2]==0)
						a.s2=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][2]==1)
						a.s2=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][2]==2)
						a.s2=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][2]==3)
						a.s2=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][2]==4)
						a.s2=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][2]==5)
						a.s2=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][2]==6)
						a.s2=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][2]==7)
						a.s2=c_add[b][7];
		
					if(c_mult[c_val[c_row_ptr[x]+y]][3]==0)
						a.s3=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][3]==1)
						a.s3=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][3]==2)
						a.s3=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][3]==3)
						a.s3=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][3]==4)
						a.s3=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][3]==5)
						a.s3=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][3]==6)
						a.s3=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][3]==7)
						a.s3=c_add[b][7];
		
					if(c_mult[c_val[c_row_ptr[x]+y]][4]==0)
						a.s4=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][4]==1)
						a.s4=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][4]==2)
						a.s4=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][4]==3)
						a.s4=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][4]==4)
						a.s4=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][4]==5)
						a.s4=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][4]==6)
						a.s4=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][4]==7)
						a.s4=c_add[b][7];
		
					if(c_mult[c_val[c_row_ptr[x]+y]][5]==0)
						a.s5=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][5]==1)
						a.s5=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][5]==2)
						a.s5=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][5]==3)
						a.s5=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][5]==4)
						a.s5=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][5]==5)
						a.s5=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][5]==6)
						a.s5=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][5]==7)
						a.s5=c_add[b][7];
		
					if(c_mult[c_val[c_row_ptr[x]+y]][6]==0)
						a.s6=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][6]==1)
						a.s6=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][6]==2)
						a.s6=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][6]==3)
						a.s6=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][6]==4)
						a.s6=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][6]==5)
						a.s6=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][6]==6)
						a.s6=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][6]==7)
						a.s6=c_add[b][7];
		
					if(c_mult[c_val[c_row_ptr[x]+y]][7]==0)
						a.s7=c_add[b][0];
					else if(c_mult[c_val[c_row_ptr[x]+y]][7]==1)
						a.s7=c_add[b][1];
					else if(c_mult[c_val[c_row_ptr[x]+y]][7]==2)
						a.s7=c_add[b][2];
					else if(c_mult[c_val[c_row_ptr[x]+y]][7]==3)
						a.s7=c_add[b][3];
					else if(c_mult[c_val[c_row_ptr[x]+y]][7]==4)
						a.s7=c_add[b][4];
					else if(c_mult[c_val[c_row_ptr[x]+y]][7]==5)
						a.s7=c_add[b][5];
					else if(c_mult[c_val[c_row_ptr[x]+y]][7]==6)
						a.s7=c_add[b][6];
					else if(c_mult[c_val[c_row_ptr[x]+y]][7]==7)
						a.s7=c_add[b][7];
					/* if(x==3){
						printf("a[%d][0]=%d\n", b, a.x);  
						printf("a[%d][1]=%d\n", b, a.y);  
						printf("a[%d][2]=%d\n", b, a.z);  
						printf("a[%d][3]=%d\n", b, a.w);  
					} */

					

					// FORWARD
					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S0
						if(a.s0==0){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s0==1){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s0==2){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s0==3){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s0==4){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s0==5){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s0==6){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s0==7){
							if(b==0)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s0 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S1
						if(a.s1==0){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s1==1){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s1==2){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s1==3){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s1==4){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s1==5){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s1==6){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s1==7){
							if(b==0)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s1 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S2
						if(a.s2==0){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s2==1){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s2==2){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s2==3){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s2==4){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s2==5){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s2==6){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s2==7){
							if(b==0)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s2= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s2 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S3
						if(a.s3==0){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s3==1){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s3==2){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s3==3){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s3==4){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s3==5){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s3==6){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s3==7){
							if(b==0)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s3= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s3 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S4
						if(a.s4==0){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s4==1){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s4==2){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s4==3){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s4==4){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s4==5){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s4==6){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s4==7){
							if(b==0)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s4= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S5
						if(a.s5==0){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s5==1){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s5==2){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s5==3){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s5==4){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s5==5){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s5==6){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s5==7){
							if(b==0)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s5= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s5 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S6
						if(a.s6==0){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s6==1){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s6==2){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s6==3){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s6==4){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s6==5){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s6==6){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s6==7){
							if(b==0)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s6= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}

					/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////77
					// S7
						if(a.s7==0){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else  if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s0;
							else if(b==7)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s0 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s0;
						}
						else if(a.s7==1){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s1;
							else if(b==7)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s1 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s1;
						}
						else if(a.s7==2){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s2;
							else if(b==7)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s2 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s2;
						}
						else if(a.s7==3){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s3;
							else if(b==7)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s3 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s3;
						}

						else if(a.s7==4){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else  if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s4;
							else if(b==7)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s4 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s4;
						}
						else if(a.s7==5){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s5;
							else if(b==7)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s5 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s5;
						}
						else if(a.s7==6){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s6;
							else if(b==7)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s6 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s6;
						}
						else if(a.s7==7){
							if(b==0)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s0) ? s_B[(threadIdx.x*c_w_row)+y+1].s0: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==1)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s1) ? s_B[(threadIdx.x*c_w_row)+y+1].s1: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==2)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s2) ? s_B[(threadIdx.x*c_w_row)+y+1].s2: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==3)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s3) ? s_B[(threadIdx.x*c_w_row)+y+1].s3: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==4)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s4) ? s_B[(threadIdx.x*c_w_row)+y+1].s4: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==5)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s5) ? s_B[(threadIdx.x*c_w_row)+y+1].s5: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==6)
								max.s7= (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s6) ? s_B[(threadIdx.x*c_w_row)+y+1].s6: s_F[(threadIdx.x*c_w_row)+y-1].s7;
							else if(b==7)
								max.s7 = (s_F[(threadIdx.x*c_w_row)+y-1].s7 < s_B[(threadIdx.x*c_w_row)+y+1].s7) ? s_B[(threadIdx.x*c_w_row)+y+1].s7: s_F[(threadIdx.x*c_w_row)+y-1].s7;
						}
					/* if(x==3){
						printf("max[%d][0]=%d\n", b, max.x);  
						printf("max[%d][1]=%d\n", b, max.y);  
						printf("max[%d][2]=%d\n", b, max.z);  
						printf("max[%d][3]=%d\n", b, max.w);  
					} */
				
					min[0].s0 = (min[0].s0 < max.s0) ? min[0].s0: max.s0;
					min[0].s1 = (min[0].s1 < max.s1) ? min[0].s1: max.s1;
					min[0].s2 = (min[0].s2 < max.s2) ? min[0].s2: max.s2;
					min[0].s3 = (min[0].s3 < max.s3) ? min[0].s3: max.s3;
					min[0].s4 = (min[0].s4 < max.s4) ? min[0].s4: max.s4;
					min[0].s5 = (min[0].s5 < max.s5) ? min[0].s5: max.s5;
					min[0].s6 = (min[0].s6 < max.s6) ? min[0].s6: max.s6;
					min[0].s7 = (min[0].s7 < max.s7) ? min[0].s7: max.s7;
				}
				reinterpret_cast<uchar8*>(d_beta_8)[(x*c_w_row)+y]=reinterpret_cast<uchar8*>(min)[0];

			}
		//}

	}

	__global__ void GPU_VN( uchar8 * d_beta_8,  uchar8 * d_alpha_8, uchar8 * d_gamma_8, int iter){
		unsigned int x=threadIdx.x+blockIdx.x*blockDim.x;
		unsigned int y=threadIdx.y+blockIdx.y*blockDim.y;
		uchar8 temp;

		temp.s0=0;
		temp.s1=0;
		temp.s2=0;
		temp.s3=0;
		temp.s4=0;
		temp.s5=0;
		temp.s6=0;
		temp.s7=0;

		#if Q==8
		__shared__ uchar8 s_beta[2*8];
		__shared__ uchar8 s_alpha_t[2*8];
		__shared__ uchar8 s_alpha_t2[2*8];
		#elif Q==16
		__shared__ unsigned char s_beta[2][32][16];
		__shared__ unsigned char s_alpha_t[2][32][16];
		__shared__ unsigned char s_alpha_t2[2][32][16];
		#elif Q==32
		__shared__ unsigned char s_beta[2][16][32];
		__shared__ unsigned char s_alpha_t[2][16][32];
		__shared__ unsigned char s_alpha_t2[2][16][32];
		#endif

		//#if Q==4
			//if(y<c_N){
		//#endif

			reinterpret_cast<uchar8*>(s_beta)[(x*c_w_row)+threadIdx.y]=reinterpret_cast<uchar8*>(d_beta_8)[(c_ptr_to_val[c_col_ptr[y]+x])];
			__syncthreads();
			/* if(threadIdx.x==1 && y==161 && iter==4){
				printf("s_beta[%d][%d][0]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s0);  
				printf("s_beta[%d][%d][1]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s1);  
				printf("s_beta[%d][%d][2]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s2);  
				printf("s_beta[%d][%d][3]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s3);
				printf("s_beta[%d][%d][4]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s4);  
				printf("s_beta[%d][%d][5]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s5);  
				printf("s_beta[%d][%d][6]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s6);  
				printf("s_beta[%d][%d][7]=%d\n", x, y, s_beta[(x*c_w_row)+threadIdx.y].s7);    
			} */


			for(unsigned char index=0; index<c_w_col; index++){
				/* if(y==297){
					printf("index[0]=%d\n",index);  
					printf("index[1]=%d\n",index);  
					printf("index[2]=%d\n",index);  
					printf("index[3]=%d\n",index);  
				}  */
				if(index!=x){
					/* if(threadIdx.x==0 && y==9){
						printf("index[0]=%d, temp=%d\n",index, temp.s0);  
						printf("index[1]=%d, temp=%d\n",index, temp.s1);  
						printf("index[2]=%d, temp=%d\n",index, temp.s2);  
						printf("index[3]=%d, temp=%d\n",index, temp.s3);
						printf("index[4]=%d, temp=%d\n",index, temp.s4);  
						printf("index[5]=%d, temp=%d\n",index, temp.s5);  
						printf("index[6]=%d, temp=%d\n",index, temp.s6);  
						printf("index[7]=%d, temp=%d\n",index, temp.s7);    
					} */
					temp.s0=temp.s0+s_beta[(index*c_w_row)+threadIdx.y].s0;
					temp.s1=temp.s1+s_beta[(index*c_w_row)+threadIdx.y].s1;
					temp.s2=temp.s2+s_beta[(index*c_w_row)+threadIdx.y].s2;
					temp.s3=temp.s3+s_beta[(index*c_w_row)+threadIdx.y].s3;
					temp.s4=temp.s4+s_beta[(index*c_w_row)+threadIdx.y].s4;
					temp.s5=temp.s5+s_beta[(index*c_w_row)+threadIdx.y].s5;
					temp.s6=temp.s6+s_beta[(index*c_w_row)+threadIdx.y].s6;
					temp.s7=temp.s7+s_beta[(index*c_w_row)+threadIdx.y].s7;
					/* if(threadIdx.x==1 && y==161 && iter==4){
						printf("temp=%d s_beta[%d][%d][0]=%d\n", temp.s0, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s0);  
						printf("temp=%d s_beta[%d][%d][1]=%d\n", temp.s1, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s1);  
						printf("temp=%d s_beta[%d][%d][2]=%d\n", temp.s2, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s2);  
						printf("temp=%d s_beta[%d][%d][3]=%d\n", temp.s3, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s3);
						printf("temp=%d s_beta[%d][%d][4]=%d\n", temp.s4, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s4);  
						printf("temp=%d s_beta[%d][%d][5]=%d\n", temp.s5, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s5);  
						printf("temp=%d s_beta[%d][%d][6]=%d\n", temp.s6, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s6);  
						printf("temp=%d s_beta[%d][%d][7]=%d\n", temp.s7, index, threadIdx.y, s_beta[(index*c_w_row)+threadIdx.y].s7);  
					} */
				}
				__syncthreads();
			}
			/* if(threadIdx.x==1 && y==161 && iter==4){
				printf("temp[%d][%d][0]=%d\n", x, y, temp.s0);  
				printf("temp[%d][%d][1]=%d\n", x, y, temp.s1);  
				printf("temp[%d][%d][2]=%d\n", x, y, temp.s2);  
				printf("temp[%d][%d][3]=%d\n", x, y, temp.s3);  
				printf("temp[%d][%d][4]=%d\n", x, y, temp.s4);  
				printf("temp[%d][%d][5]=%d\n", x, y, temp.s5);  
				printf("temp[%d][%d][6]=%d\n", x, y, temp.s6);  
				printf("temp[%d][%d][7]=%d\n", x, y, temp.s7);  
			}  */

			s_alpha_t[(x*c_w_row)+threadIdx.y].s0= temp.s0+ d_gamma_8[y].s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s1= temp.s1+ d_gamma_8[y].s1;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s2= temp.s2+ d_gamma_8[y].s2;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s3= temp.s3+ d_gamma_8[y].s3;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s4= temp.s4+ d_gamma_8[y].s4;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s5= temp.s5+ d_gamma_8[y].s5;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s6= temp.s6+ d_gamma_8[y].s6;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s7= temp.s7+ d_gamma_8[y].s7;
			/* if(threadIdx.x==1 && y==161 && iter==4){
				printf("s_alpha_t[%d][%d][0]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s0);  
				printf("s_alpha_t[%d][%d][1]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s1);  
				printf("s_alpha_t[%d][%d][2]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s2);  
				printf("s_alpha_t[%d][%d][3]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s3); 
				printf("s_alpha_t[%d][%d][4]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s4);  
				printf("s_alpha_t[%d][%d][5]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s5);  
				printf("s_alpha_t[%d][%d][6]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s6);  
				printf("s_alpha_t[%d][%d][7]=%d\n", x, y, s_alpha_t[(x*c_w_row)+threadIdx.y].s7); 
			} */

			s_alpha_t2[(x*c_w_row)+threadIdx.y].s0=0;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s1=1;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s2=2;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s3=3;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s4=4;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s5=5;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s6=6;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s7=7;
			__syncthreads();


			s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s0 > s_alpha_t[(x*c_w_row)+threadIdx.y].s4) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s4 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s0;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s1= (s_alpha_t[(x*c_w_row)+threadIdx.y].s1 > s_alpha_t[(x*c_w_row)+threadIdx.y].s5) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s5 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s1;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s2= (s_alpha_t[(x*c_w_row)+threadIdx.y].s2 > s_alpha_t[(x*c_w_row)+threadIdx.y].s6) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s6 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s2;
			s_alpha_t2[(x*c_w_row)+threadIdx.y].s3= (s_alpha_t[(x*c_w_row)+threadIdx.y].s3 > s_alpha_t[(x*c_w_row)+threadIdx.y].s7) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s7 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s3;
			/* if(threadIdx.x==1 && y==161 && iter==4){
				printf("s_alpha_t2[%d][%d][0]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s0);
				printf("s_alpha_t2[%d][%d][1]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s1);
				printf("s_alpha_t2[%d][%d][2]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s2);
				printf("s_alpha_t2[%d][%d][3]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s3); 
				printf("s_alpha_t2[%d][%d][4]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s4);
				printf("s_alpha_t2[%d][%d][5]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s5);
				printf("s_alpha_t2[%d][%d][6]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s6);
				printf("s_alpha_t2[%d][%d][7]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s7);  
			} */

			if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==0){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s2==2)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s0 > s_alpha_t[(x*c_w_row)+threadIdx.y].s2) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s2 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s0;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s2==6)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s0 > s_alpha_t[(x*c_w_row)+threadIdx.y].s6) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s6 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s0;
			}
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==4){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s2==2)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s4 > s_alpha_t[(x*c_w_row)+threadIdx.y].s2) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s2 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s4;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s2==6)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s4 > s_alpha_t[(x*c_w_row)+threadIdx.y].s6) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s6 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s4;
			}

			if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==1){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s3==3)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s1= (s_alpha_t[(x*c_w_row)+threadIdx.y].s1 > s_alpha_t[(x*c_w_row)+threadIdx.y].s3) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s3 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s1;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s3==7)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s1= (s_alpha_t[(x*c_w_row)+threadIdx.y].s1 > s_alpha_t[(x*c_w_row)+threadIdx.y].s7) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s7 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s1;
			}
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==5){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s3==3)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s1= (s_alpha_t[(x*c_w_row)+threadIdx.y].s5 > s_alpha_t[(x*c_w_row)+threadIdx.y].s3) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s3 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s5;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s3==7)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s1= (s_alpha_t[(x*c_w_row)+threadIdx.y].s5 > s_alpha_t[(x*c_w_row)+threadIdx.y].s7) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s7 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s5;
			}
			/* if(threadIdx.x==1 && y==161 && iter==4){
				printf("s_alpha_t2[%d][%d][0]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s0);
				printf("s_alpha_t2[%d][%d][1]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s1);
				printf("s_alpha_t2[%d][%d][2]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s2);
				printf("s_alpha_t2[%d][%d][3]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s3); 
				printf("s_alpha_t2[%d][%d][4]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s4);
				printf("s_alpha_t2[%d][%d][5]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s5);
				printf("s_alpha_t2[%d][%d][6]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s6);
				printf("s_alpha_t2[%d][%d][7]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s7);  
			}  */
			


			if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==0){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==1)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s0 > s_alpha_t[(x*c_w_row)+threadIdx.y].s1) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s1 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s0;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==3)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s0 > s_alpha_t[(x*c_w_row)+threadIdx.y].s3) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s3 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s0;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==5)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s0 > s_alpha_t[(x*c_w_row)+threadIdx.y].s5) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s5 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s0;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==7)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s0 > s_alpha_t[(x*c_w_row)+threadIdx.y].s7) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s7 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s0;
			}
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==2){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==1)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s2 > s_alpha_t[(x*c_w_row)+threadIdx.y].s1) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s1 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s2;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==3)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s2 > s_alpha_t[(x*c_w_row)+threadIdx.y].s3) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s3 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s2;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==5)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s2 > s_alpha_t[(x*c_w_row)+threadIdx.y].s5) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s5 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s2;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==7)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s2 > s_alpha_t[(x*c_w_row)+threadIdx.y].s7) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s7 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s2;
			}
			if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==4){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==1)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s4 > s_alpha_t[(x*c_w_row)+threadIdx.y].s1) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s1 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s4;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==3)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s4 > s_alpha_t[(x*c_w_row)+threadIdx.y].s3) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s3 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s4;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==5)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s4 > s_alpha_t[(x*c_w_row)+threadIdx.y].s5) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s5 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s4;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==7)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s4 > s_alpha_t[(x*c_w_row)+threadIdx.y].s7) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s7 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s4;
			}
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==6){
				if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==1)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s6 > s_alpha_t[(x*c_w_row)+threadIdx.y].s1) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s1 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s6;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==3)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s6 > s_alpha_t[(x*c_w_row)+threadIdx.y].s3) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s3 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s6;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==5)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s6 > s_alpha_t[(x*c_w_row)+threadIdx.y].s5) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s5 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s6;
				else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s1==7)
					s_alpha_t2[(x*c_w_row)+threadIdx.y].s0= (s_alpha_t[(x*c_w_row)+threadIdx.y].s6 > s_alpha_t[(x*c_w_row)+threadIdx.y].s7) ? s_alpha_t2[(x*c_w_row)+threadIdx.y].s7 : s_alpha_t2[(x*c_w_row)+threadIdx.y].s6;
			}
			__syncthreads();
			/* if(threadIdx.x==1 && y==161 && iter==4){
				printf("s_alpha_t2[%d][%d][0]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s0);
				printf("s_alpha_t2[%d][%d][1]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s1);
				printf("s_alpha_t2[%d][%d][2]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s2);
				printf("s_alpha_t2[%d][%d][3]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s3); 
				printf("s_alpha_t2[%d][%d][4]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s4);
				printf("s_alpha_t2[%d][%d][5]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s5);
				printf("s_alpha_t2[%d][%d][6]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s6);
				printf("s_alpha_t2[%d][%d][7]=%d\n", x, y, s_alpha_t2[(x*c_w_row)+threadIdx.y].s7);  
			} */
			
			if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==0)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s0;
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==1)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s1;
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==2)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s2;
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==3)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s3;
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==4)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s4;
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==5)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s5;
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==6)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s6;
			else if(s_alpha_t2[(x*c_w_row)+threadIdx.y].s0==7)
				temp.s0=s_alpha_t[(x*c_w_row)+threadIdx.y].s7;

			s_alpha_t[(x*c_w_row)+threadIdx.y].s0 =s_alpha_t[(x*c_w_row)+threadIdx.y].s0 - temp.s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s1 =s_alpha_t[(x*c_w_row)+threadIdx.y].s1 - temp.s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s2 =s_alpha_t[(x*c_w_row)+threadIdx.y].s2 - temp.s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s3 =s_alpha_t[(x*c_w_row)+threadIdx.y].s3 - temp.s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s4 =s_alpha_t[(x*c_w_row)+threadIdx.y].s4 - temp.s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s5 =s_alpha_t[(x*c_w_row)+threadIdx.y].s5 - temp.s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s6 =s_alpha_t[(x*c_w_row)+threadIdx.y].s6 - temp.s0;
			s_alpha_t[(x*c_w_row)+threadIdx.y].s7 =s_alpha_t[(x*c_w_row)+threadIdx.y].s7 - temp.s0;


			reinterpret_cast<uchar8*>(d_alpha_8)[(c_ptr_to_val[c_col_ptr[y]+x])]=reinterpret_cast<uchar8*>(s_alpha_t)[(x*c_w_row)+threadIdx.y];
			/* if(y==161 && iter==4){
				printf("d_alpha_8[%d][0]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s0);
				printf("d_alpha_8[%d][1]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s1);
				printf("d_alpha_8[%d][2]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s2);
				printf("d_alpha_8[%d][3]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s3);  
				printf("d_alpha_8[%d][4]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s4);
				printf("d_alpha_8[%d][5]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s5);
				printf("d_alpha_8[%d][6]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s6);
				printf("d_alpha_8[%d][7]=%d\n", (c_ptr_to_val[c_col_ptr[y]+x]), d_alpha_8[(c_ptr_to_val[c_col_ptr[y]+x])].s7); 
			} */
	}
//#endif



//===================================
// CUDA Wrapper
//===================================

extern "C" int cuda_minmax(unsigned char* h_beta, unsigned char* ALPHAmn_, unsigned char* GAMMAn_, unsigned char* h_F,unsigned char* h_B,int* iteration){
	
	#if Q==4
		const unsigned char add[4][4] =	{	{ 0,1,2,3 },
										{ 1,0,3,2 },
										{ 2,3,0,1 },
										{ 3,2,1,0 }};
		const unsigned char mult[4][4] = {{ 0,0,0,0}, { 0,1,2,3 },{ 0,2,3,1 },{ 0,3,1,2 }};
		const unsigned char inv[4] = { 0,1,3,2};
	#elif Q==8
		const unsigned char add[8][8] = {{ 0,1,2,3,4,5,6,7 }, { 1,0,3,2,5,4,7,6 },{ 2,3,0,1,6,7,4,5 },{ 3,2,1,0,7,6,5,4 },{ 4,5,6,7,0,1,2,3 }, { 5,4,7,6,1,0,3,2 },{ 6,7,4,5,2,3,0,1 },{ 7,6,5,4,3,2,1,0 }};
		const unsigned char mult[8][8] = {{ 0,0,0,0,0,0,0,0}, { 0,1,2,3,4,5,6,7 },{ 0,2,4,6,3,1,7,5 },{ 0,3,6,5,7,4,1,2 },{ 0,4,3,7,6,2,5,1}, { 0,5,1,4,2,7,3,6 },{ 0,6,7,1,5,3,2,4 },{ 0,7,5,2,1,6,4,3 }};
		const unsigned char inv[8] = { 0,1,5,6,7,2,3,4};
	#elif Q==16
		const unsigned char add[16][16] = {	{ 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15},
											{ 1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14 },
											{ 2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13 },
											{ 3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12 },
											{ 4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11 },
											{ 5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10 },
											{ 6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9 },
											{ 7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8 },
											{ 8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7 },
											{ 9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6 },
											{ 10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5 },
											{ 11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4 },
											{ 12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3 },
											{ 13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2 },
											{ 14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1 },
											{ 15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0 }};

		const unsigned char mult[16][16] = {{ 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
											{ 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15 },
											{ 0,2,4,6,8,10,12,14,3,1,7,5,11,9,15,13 },
											{ 0,3,6,5,12,15,10,9,11,8,13,14,7,4,1,2 },
											{ 0,4,8,12,3,7,11,15,6,2,14,10,5,1,13,9},
											{ 0,5,10,15,7,2,13,8,14,11,4,1,9,12,3,6},
											{ 0,6,12,10,11,13,7,1,5,3,9,15,14,8,2,4},
											{ 0,7,14,9,15,8,1,6,13,10,3,4,2,5,12,11},
											{ 0,8,3,11,6,14,5,13,12,4,15,7,10,2,9,1},
											{ 0,9,1,8,2,11,3,10,4,13,5,12,6,15,7,14},
											{ 0,10,7,13,14,4,9,3,15,5,8,2,1,11,6,12},
											{ 0,11,5,14,10,1,15,4,7,12,2,9,13,6,8,3},
											{ 0,12,11,7,5,9,14,2,10,6,1,13,15,3,4,8},
											{ 0,13,9,4,1,12,8,5,2,15,11,6,3,14,10,7},
											{ 0,14,15,1,13,3,2,12,9,7,6,8,4,10,11,5},
											{ 0,15,13,2,9,6,4,11,1,14,12,3,8,7,5,10}};

		const unsigned char inv[16] = { 0,1,9,14,13,11,7,6,15,2,12,5,10,4,3,8};
	#elif Q==32
		const unsigned char add[32][32] = {	{0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31},
											{1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14,17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30},
											{2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13,18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29},
											{3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12,19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28},
											{4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11,20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27},
											{5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10,21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26},
											{6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9,22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25},
											{7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8,23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24},
											{8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7,24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23},
											{9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6,25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22},
											{10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5,26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21},
											{11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4,27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20},
											{12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3,28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19},
											{13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2,29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18},
											{14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1,30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17},
											{15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0,31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16},
											{16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15},
											{17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30,1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14},
											{18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29,2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13},
											{19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28,3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12},
											{20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27,4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11},
											{21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26,5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10},
											{22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25,6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9},
											{23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24,7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8},
											{24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23,8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7},
											{25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22,9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6},
											{26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21,10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5},
											{27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20,11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4},
											{28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19,12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3},
											{29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18,13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2},
											{30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17,14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1},
											{31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0}};


		const unsigned char mult[32][32] = {{0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
											{0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31},
											{0,2,4,6,8,10,12,14,16,18,20,22,24,26,28,30,5,7,1,3,13,15,9,11,21,23,17,19,29,31,25,27},
											{0,3,6,5,12,15,10,9,24,27,30,29,20,23,18,17,21,22,19,16,25,26,31,28,13,14,11,8,1,2,7,4},
											{0,4,8,12,16,20,24,28,5,1,13,9,21,17,29,25,10,14,2,6,26,30,18,22,15,11,7,3,31,27,23,19},
											{0,5,10,15,20,17,30,27,13,8,7,2,25,28,19,22,26,31,16,21,14,11,4,1,23,18,29,24,3,6,9,12},
											{0,6,12,10,24,30,20,18,21,19,25,31,13,11,1,7,15,9,3,5,23,17,27,29,26,28,22,16,2,4,14,8},
											{0,7,14,9,28,27,18,21,29,26,19,20,1,6,15,8,31,24,17,22,3,4,13,10,2,5,12,11,30,25,16,23},
											{0,8,16,24,5,13,21,29,10,2,26,18,15,7,31,23,20,28,4,12,17,25,1,9,30,22,14,6,27,19,11,3},
											{0,9,18,27,1,8,19,26,2,11,16,25,3,10,17,24,4,13,22,31,5,12,23,30,6,15,20,29,7,14,21,28},
											{0,10,20,30,13,7,25,19,26,16,14,4,23,29,3,9,17,27,5,15,28,22,8,2,11,1,31,21,6,12,18,24},
											{0,11,22,29,9,2,31,20,18,25,4,15,27,16,13,6,1,10,23,28,8,3,30,21,19,24,5,14,26,17,12,7},
											{0,12,24,20,21,25,13,1,15,3,23,27,26,22,2,14,30,18,6,10,11,7,19,31,17,29,9,5,4,8,28,16},
											{0,13,26,23,17,28,11,6,7,10,29,16,22,27,12,1,14,3,20,25,31,18,5,8,9,4,19,30,24,21,2,15},
											{0,14,28,18,29,19,1,15,31,17,3,13,2,12,30,16,27,21,7,9,6,8,26,20,4,10,24,22,25,23,5,11},
											{0,15,30,17,25,22,7,8,23,24,9,6,14,1,16,31,11,4,21,26,18,29,12,3,28,19,2,13,5,10,27,20},
											{0,16,5,21,10,26,15,31,20,4,17,1,30,14,27,11,13,29,8,24,7,23,2,18,25,9,28,12,19,3,22,6},
											{0,17,7,22,14,31,9,24,28,13,27,10,18,3,21,4,29,12,26,11,19,2,20,5,1,16,6,23,15,30,8,25},
											{0,18,1,19,2,16,3,17,4,22,5,23,6,20,7,21,8,26,9,27,10,24,11,25,12,30,13,31,14,28,15,29},
											{0,19,3,16,6,21,5,22,12,31,15,28,10,25,9,26,24,11,27,8,30,13,29,14,20,7,23,4,18,1,17,2},
											{0,20,13,25,26,14,23,3,17,5,28,8,11,31,6,18,7,19,10,30,29,9,16,4,22,2,27,15,12,24,1,21},
											{0,21,15,26,30,11,17,4,25,12,22,3,7,18,8,29,23,2,24,13,9,28,6,19,14,27,1,20,16,5,31,10},
											{0,22,9,31,18,4,27,13,1,23,8,30,19,5,26,12,2,20,11,29,16,6,25,15,3,21,10,28,17,7,24,14},
											{0,23,11,28,22,1,29,10,9,30,2,21,31,8,20,3,18,5,25,14,4,19,15,24,27,12,16,7,13,26,6,17},
											{0,24,21,13,15,23,26,2,30,6,11,19,17,9,4,28,25,1,12,20,22,14,3,27,7,31,18,10,8,16,29,5},
											{0,25,23,14,11,18,28,5,22,15,1,24,29,4,10,19,9,16,30,7,2,27,21,12,31,6,8,17,20,13,3,26},
											{0,26,17,11,7,29,22,12,14,20,31,5,9,19,24,2,28,6,13,23,27,1,10,16,18,8,3,25,21,15,4,30},
											{0,27,19,8,3,24,16,11,6,29,21,14,5,30,22,13,12,23,31,4,15,20,28,7,10,17,25,2,9,18,26,1},
											{0,28,29,1,31,3,2,30,27,7,6,26,4,24,25,5,19,15,14,18,12,16,17,13,8,20,21,9,23,11,10,22},
											{0,29,31,2,27,6,4,25,19,14,12,17,8,21,23,10,3,30,28,1,24,5,7,26,16,13,15,18,11,22,20,9},
											{0,30,25,7,23,9,14,16,11,21,18,12,28,2,5,27,22,8,15,17,1,31,24,6,29,3,4,26,10,20,19,13},
											{0,31,27,4,19,12,8,23,3,28,24,7,16,15,11,20,6,25,29,2,21,10,14,17,5,26,30,1,22,9,13,18}};
	
		const unsigned char inv[32] = { 0,1,18,28,9,23,14,12,22,4,25,16,7,15,6,13,11,24,2,29,30,26,8,5,17,10,21,31,3,19,20,27};
		
	#elif Q==64
		const unsigned char add[64][64] = {{0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39,40,41,42,43,44,45,46,47,48,49,50,51,52,53,54,55,56,57,58,59,60,61,62,63},
		{1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14,17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30,33,32,35,34,37,36,39,38,41,40,43,42,45,44,47,46,49,48,51,50,53,52,55,54,57,56,59,58,61,60,63,62},
		{2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13,18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29,34,35,32,33,38,39,36,37,42,43,40,41,46,47,44,45,50,51,48,49,54,55,52,53,58,59,56,57,62,63,60,61},
		{3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12,19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28,35,34,33,32,39,38,37,36,43,42,41,40,47,46,45,44,51,50,49,48,55,54,53,52,59,58,57,56,63,62,61,60},
		{4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11,20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27,36,37,38,39,32,33,34,35,44,45,46,47,40,41,42,43,52,53,54,55,48,49,50,51,60,61,62,63,56,57,58,59},
		{5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10,21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26,37,36,39,38,33,32,35,34,45,44,47,46,41,40,43,42,53,52,55,54,49,48,51,50,61,60,63,62,57,56,59,58},
		{6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9,22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25,38,39,36,37,34,35,32,33,46,47,44,45,42,43,40,41,54,55,52,53,50,51,48,49,62,63,60,61,58,59,56,57},
		{7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8,23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24,39,38,37,36,35,34,33,32,47,46,45,44,43,42,41,40,55,54,53,52,51,50,49,48,63,62,61,60,59,58,57,56},
		{8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7,24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23,40,41,42,43,44,45,46,47,32,33,34,35,36,37,38,39,56,57,58,59,60,61,62,63,48,49,50,51,52,53,54,55},
		{9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6,25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22,41,40,43,42,45,44,47,46,33,32,35,34,37,36,39,38,57,56,59,58,61,60,63,62,49,48,51,50,53,52,55,54},
		{10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5,26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21,42,43,40,41,46,47,44,45,34,35,32,33,38,39,36,37,58,59,56,57,62,63,60,61,50,51,48,49,54,55,52,53},
		{11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4,27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20,43,42,41,40,47,46,45,44,35,34,33,32,39,38,37,36,59,58,57,56,63,62,61,60,51,50,49,48,55,54,53,52},
		{12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3,28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19,44,45,46,47,40,41,42,43,36,37,38,39,32,33,34,35,60,61,62,63,56,57,58,59,52,53,54,55,48,49,50,51},
		{13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2,29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18,45,44,47,46,41,40,43,42,37,36,39,38,33,32,35,34,61,60,63,62,57,56,59,58,53,52,55,54,49,48,51,50},
		{14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1,30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17,46,47,44,45,42,43,40,41,38,39,36,37,34,35,32,33,62,63,60,61,58,59,56,57,54,55,52,53,50,51,48,49},
		{15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0,31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16,47,46,45,44,43,42,41,40,39,38,37,36,35,34,33,32,63,62,61,60,59,58,57,56,55,54,53,52,51,50,49,48},
		{16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,48,49,50,51,52,53,54,55,56,57,58,59,60,61,62,63,32,33,34,35,36,37,38,39,40,41,42,43,44,45,46,47},
		{17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30,1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14,49,48,51,50,53,52,55,54,57,56,59,58,61,60,63,62,33,32,35,34,37,36,39,38,41,40,43,42,45,44,47,46},
		{18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29,2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13,50,51,48,49,54,55,52,53,58,59,56,57,62,63,60,61,34,35,32,33,38,39,36,37,42,43,40,41,46,47,44,45},
		{19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28,3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12,51,50,49,48,55,54,53,52,59,58,57,56,63,62,61,60,35,34,33,32,39,38,37,36,43,42,41,40,47,46,45,44},
		{20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27,4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11,52,53,54,55,48,49,50,51,60,61,62,63,56,57,58,59,36,37,38,39,32,33,34,35,44,45,46,47,40,41,42,43},
		{21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26,5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10,53,52,55,54,49,48,51,50,61,60,63,62,57,56,59,58,37,36,39,38,33,32,35,34,45,44,47,46,41,40,43,42},
		{22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25,6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9,54,55,52,53,50,51,48,49,62,63,60,61,58,59,56,57,38,39,36,37,34,35,32,33,46,47,44,45,42,43,40,41},
		{23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24,7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8,55,54,53,52,51,50,49,48,63,62,61,60,59,58,57,56,39,38,37,36,35,34,33,32,47,46,45,44,43,42,41,40},
		{24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23,8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7,56,57,58,59,60,61,62,63,48,49,50,51,52,53,54,55,40,41,42,43,44,45,46,47,32,33,34,35,36,37,38,39},
		{25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22,9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6,57,56,59,58,61,60,63,62,49,48,51,50,53,52,55,54,41,40,43,42,45,44,47,46,33,32,35,34,37,36,39,38},
		{26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21,10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5,58,59,56,57,62,63,60,61,50,51,48,49,54,55,52,53,42,43,40,41,46,47,44,45,34,35,32,33,38,39,36,37},
		{27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20,11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4,59,58,57,56,63,62,61,60,51,50,49,48,55,54,53,52,43,42,41,40,47,46,45,44,35,34,33,32,39,38,37,36},
		{28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19,12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3,60,61,62,63,56,57,58,59,52,53,54,55,48,49,50,51,44,45,46,47,40,41,42,43,36,37,38,39,32,33,34,35},
		{29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18,13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2,61,60,63,62,57,56,59,58,53,52,55,54,49,48,51,50,45,44,47,46,41,40,43,42,37,36,39,38,33,32,35,34},
		{30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17,14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1,62,63,60,61,58,59,56,57,54,55,52,53,50,51,48,49,46,47,44,45,42,43,40,41,38,39,36,37,34,35,32,33},
		{31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0,63,62,61,60,59,58,57,56,55,54,53,52,51,50,49,48,47,46,45,44,43,42,41,40,39,38,37,36,35,34,33,32},
		{32,33,34,35,36,37,38,39,40,41,42,43,44,45,46,47,48,49,50,51,52,53,54,55,56,57,58,59,60,61,62,63,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31},
		{33,32,35,34,37,36,39,38,41,40,43,42,45,44,47,46,49,48,51,50,53,52,55,54,57,56,59,58,61,60,63,62,1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14,17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30},
		{34,35,32,33,38,39,36,37,42,43,40,41,46,47,44,45,50,51,48,49,54,55,52,53,58,59,56,57,62,63,60,61,2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13,18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29},
		{35,34,33,32,39,38,37,36,43,42,41,40,47,46,45,44,51,50,49,48,55,54,53,52,59,58,57,56,63,62,61,60,3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12,19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28},
		{36,37,38,39,32,33,34,35,44,45,46,47,40,41,42,43,52,53,54,55,48,49,50,51,60,61,62,63,56,57,58,59,4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11,20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27},
		{37,36,39,38,33,32,35,34,45,44,47,46,41,40,43,42,53,52,55,54,49,48,51,50,61,60,63,62,57,56,59,58,5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10,21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26},
		{38,39,36,37,34,35,32,33,46,47,44,45,42,43,40,41,54,55,52,53,50,51,48,49,62,63,60,61,58,59,56,57,6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9,22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25},
		{39,38,37,36,35,34,33,32,47,46,45,44,43,42,41,40,55,54,53,52,51,50,49,48,63,62,61,60,59,58,57,56,7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8,23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24},
		{40,41,42,43,44,45,46,47,32,33,34,35,36,37,38,39,56,57,58,59,60,61,62,63,48,49,50,51,52,53,54,55,8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7,24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23},
		{41,40,43,42,45,44,47,46,33,32,35,34,37,36,39,38,57,56,59,58,61,60,63,62,49,48,51,50,53,52,55,54,9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6,25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22},
		{42,43,40,41,46,47,44,45,34,35,32,33,38,39,36,37,58,59,56,57,62,63,60,61,50,51,48,49,54,55,52,53,10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5,26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21},
		{43,42,41,40,47,46,45,44,35,34,33,32,39,38,37,36,59,58,57,56,63,62,61,60,51,50,49,48,55,54,53,52,11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4,27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20},
		{44,45,46,47,40,41,42,43,36,37,38,39,32,33,34,35,60,61,62,63,56,57,58,59,52,53,54,55,48,49,50,51,12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3,28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19},
		{45,44,47,46,41,40,43,42,37,36,39,38,33,32,35,34,61,60,63,62,57,56,59,58,53,52,55,54,49,48,51,50,13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2,29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18},
		{46,47,44,45,42,43,40,41,38,39,36,37,34,35,32,33,62,63,60,61,58,59,56,57,54,55,52,53,50,51,48,49,14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1,30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17},
		{47,46,45,44,43,42,41,40,39,38,37,36,35,34,33,32,63,62,61,60,59,58,57,56,55,54,53,52,51,50,49,48,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0,31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16},
		{48,49,50,51,52,53,54,55,56,57,58,59,60,61,62,63,32,33,34,35,36,37,38,39,40,41,42,43,44,45,46,47,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15},
		{49,48,51,50,53,52,55,54,57,56,59,58,61,60,63,62,33,32,35,34,37,36,39,38,41,40,43,42,45,44,47,46,17,16,19,18,21,20,23,22,25,24,27,26,29,28,31,30,1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14},
		{50,51,48,49,54,55,52,53,58,59,56,57,62,63,60,61,34,35,32,33,38,39,36,37,42,43,40,41,46,47,44,45,18,19,16,17,22,23,20,21,26,27,24,25,30,31,28,29,2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13},
		{51,50,49,48,55,54,53,52,59,58,57,56,63,62,61,60,35,34,33,32,39,38,37,36,43,42,41,40,47,46,45,44,19,18,17,16,23,22,21,20,27,26,25,24,31,30,29,28,3,2,1,0,7,6,5,4,11,10,9,8,15,14,13,12},
		{52,53,54,55,48,49,50,51,60,61,62,63,56,57,58,59,36,37,38,39,32,33,34,35,44,45,46,47,40,41,42,43,20,21,22,23,16,17,18,19,28,29,30,31,24,25,26,27,4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11},
		{53,52,55,54,49,48,51,50,61,60,63,62,57,56,59,58,37,36,39,38,33,32,35,34,45,44,47,46,41,40,43,42,21,20,23,22,17,16,19,18,29,28,31,30,25,24,27,26,5,4,7,6,1,0,3,2,13,12,15,14,9,8,11,10},
		{54,55,52,53,50,51,48,49,62,63,60,61,58,59,56,57,38,39,36,37,34,35,32,33,46,47,44,45,42,43,40,41,22,23,20,21,18,19,16,17,30,31,28,29,26,27,24,25,6,7,4,5,2,3,0,1,14,15,12,13,10,11,8,9},
		{55,54,53,52,51,50,49,48,63,62,61,60,59,58,57,56,39,38,37,36,35,34,33,32,47,46,45,44,43,42,41,40,23,22,21,20,19,18,17,16,31,30,29,28,27,26,25,24,7,6,5,4,3,2,1,0,15,14,13,12,11,10,9,8},
		{56,57,58,59,60,61,62,63,48,49,50,51,52,53,54,55,40,41,42,43,44,45,46,47,32,33,34,35,36,37,38,39,24,25,26,27,28,29,30,31,16,17,18,19,20,21,22,23,8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7},
		{57,56,59,58,61,60,63,62,49,48,51,50,53,52,55,54,41,40,43,42,45,44,47,46,33,32,35,34,37,36,39,38,25,24,27,26,29,28,31,30,17,16,19,18,21,20,23,22,9,8,11,10,13,12,15,14,1,0,3,2,5,4,7,6},
		{58,59,56,57,62,63,60,61,50,51,48,49,54,55,52,53,42,43,40,41,46,47,44,45,34,35,32,33,38,39,36,37,26,27,24,25,30,31,28,29,18,19,16,17,22,23,20,21,10,11,8,9,14,15,12,13,2,3,0,1,6,7,4,5},
		{59,58,57,56,63,62,61,60,51,50,49,48,55,54,53,52,43,42,41,40,47,46,45,44,35,34,33,32,39,38,37,36,27,26,25,24,31,30,29,28,19,18,17,16,23,22,21,20,11,10,9,8,15,14,13,12,3,2,1,0,7,6,5,4},
		{60,61,62,63,56,57,58,59,52,53,54,55,48,49,50,51,44,45,46,47,40,41,42,43,36,37,38,39,32,33,34,35,28,29,30,31,24,25,26,27,20,21,22,23,16,17,18,19,12,13,14,15,8,9,10,11,4,5,6,7,0,1,2,3},
		{61,60,63,62,57,56,59,58,53,52,55,54,49,48,51,50,45,44,47,46,41,40,43,42,37,36,39,38,33,32,35,34,29,28,31,30,25,24,27,26,21,20,23,22,17,16,19,18,13,12,15,14,9,8,11,10,5,4,7,6,1,0,3,2},
		{62,63,60,61,58,59,56,57,54,55,52,53,50,51,48,49,46,47,44,45,42,43,40,41,38,39,36,37,34,35,32,33,30,31,28,29,26,27,24,25,22,23,20,21,18,19,16,17,14,15,12,13,10,11,8,9,6,7,4,5,2,3,0,1},
		{63,62,61,60,59,58,57,56,55,54,53,52,51,50,49,48,47,46,45,44,43,42,41,40,39,38,37,36,35,34,33,32,31,30,29,28,27,26,25,24,23,22,21,20,19,18,17,16,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0}
		};


		const unsigned char mult[64][64] = {{0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
		{0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39,40,41,42,43,44,45,46,47,48,49,50,51,52,53,54,55,56,57,58,59,60,61,62,63},
		{0,2,4,6,8,10,12,14,16,18,20,22,24,26,28,30,32,34,36,38,40,42,44,46,48,50,52,54,56,58,60,62,3,1,7,5,11,9,15,13,19,17,23,21,27,25,31,29,35,33,39,37,43,41,47,45,51,49,55,53,59,57,63,61},
		{0,3,6,5,12,15,10,9,24,27,30,29,20,23,18,17,48,51,54,53,60,63,58,57,40,43,46,45,36,39,34,33,35,32,37,38,47,44,41,42,59,56,61,62,55,52,49,50,19,16,21,22,31,28,25,26,11,8,13,14,7,4,1,2},
		{0,4,8,12,16,20,24,28,32,36,40,44,48,52,56,60,3,7,11,15,19,23,27,31,35,39,43,47,51,55,59,63,6,2,14,10,22,18,30,26,38,34,46,42,54,50,62,58,5,1,13,9,21,17,29,25,37,33,45,41,53,49,61,57},
		{0,5,10,15,20,17,30,27,40,45,34,39,60,57,54,51,19,22,25,28,7,2,13,8,59,62,49,52,47,42,37,32,38,35,44,41,50,55,56,61,14,11,4,1,26,31,16,21,53,48,63,58,33,36,43,46,29,24,23,18,9,12,3,6},
		{0,6,12,10,24,30,20,18,48,54,60,58,40,46,36,34,35,37,47,41,59,61,55,49,19,21,31,25,11,13,7,1,5,3,9,15,29,27,17,23,53,51,57,63,45,43,33,39,38,32,42,44,62,56,50,52,22,16,26,28,14,8,2,4},
		{0,7,14,9,28,27,18,21,56,63,54,49,36,35,42,45,51,52,61,58,47,40,33,38,11,12,5,2,23,16,25,30,37,34,43,44,57,62,55,48,29,26,19,20,1,6,15,8,22,17,24,31,10,13,4,3,46,41,32,39,50,53,60,59},
		{0,8,16,24,32,40,48,56,3,11,19,27,35,43,51,59,6,14,22,30,38,46,54,62,5,13,21,29,37,45,53,61,12,4,28,20,44,36,60,52,15,7,31,23,47,39,63,55,10,2,26,18,42,34,58,50,9,1,25,17,41,33,57,49},
		{0,9,18,27,36,45,54,63,11,2,25,16,47,38,61,52,22,31,4,13,50,59,32,41,29,20,15,6,57,48,43,34,44,37,62,55,8,1,26,19,39,46,53,60,3,10,17,24,58,51,40,33,30,23,12,5,49,56,35,42,21,28,7,14},
		{0,10,20,30,40,34,60,54,19,25,7,13,59,49,47,37,38,44,50,56,14,4,26,16,53,63,33,43,29,23,9,3,15,5,27,17,39,45,51,57,28,22,8,2,52,62,32,42,41,35,61,55,1,11,21,31,58,48,46,36,18,24,6,12},
		{0,11,22,29,44,39,58,49,27,16,13,6,55,60,33,42,54,61,32,43,26,17,12,7,45,38,59,48,1,10,23,28,47,36,57,50,3,8,21,30,52,63,34,41,24,19,14,5,25,18,15,4,53,62,35,40,2,9,20,31,46,37,56,51},
		{0,12,24,20,48,60,40,36,35,47,59,55,19,31,11,7,5,9,29,17,53,57,45,33,38,42,62,50,22,26,14,2,10,6,18,30,58,54,34,46,41,37,49,61,25,21,1,13,15,3,23,27,63,51,39,43,44,32,52,56,28,16,4,8},
		{0,13,26,23,52,57,46,35,43,38,49,60,31,18,5,8,21,24,15,2,33,44,59,54,62,51,36,41,10,7,16,29,42,39,48,61,30,19,4,9,1,12,27,22,53,56,47,34,63,50,37,40,11,6,17,28,20,25,14,3,32,45,58,55},
		{0,14,28,18,56,54,36,42,51,61,47,33,11,5,23,25,37,43,57,55,29,19,1,15,22,24,10,4,46,32,50,60,9,7,21,27,49,63,45,35,58,52,38,40,2,12,30,16,44,34,48,62,20,26,8,6,31,17,3,13,39,41,59,53},
		{0,15,30,17,60,51,34,45,59,52,37,42,7,8,25,22,53,58,43,36,9,6,23,24,14,1,16,31,50,61,44,35,41,38,55,56,21,26,11,4,18,29,12,3,46,33,48,63,28,19,2,13,32,47,62,49,39,40,57,54,27,20,5,10},
		{0,16,32,48,3,19,35,51,6,22,38,54,5,21,37,53,12,28,44,60,15,31,47,63,10,26,42,58,9,25,41,57,24,8,56,40,27,11,59,43,30,14,62,46,29,13,61,45,20,4,52,36,23,7,55,39,18,2,50,34,17,1,49,33},
		{0,17,34,51,7,22,37,52,14,31,44,61,9,24,43,58,28,13,62,47,27,10,57,40,18,3,48,33,21,4,55,38,56,41,26,11,63,46,29,12,54,39,20,5,49,32,19,2,36,53,6,23,35,50,1,16,42,59,8,25,45,60,15,30},
		{0,18,36,54,11,25,47,61,22,4,50,32,29,15,57,43,44,62,8,26,39,53,3,17,58,40,30,12,49,35,21,7,27,9,63,45,16,2,52,38,13,31,41,59,6,20,34,48,55,37,19,1,60,46,24,10,33,51,5,23,42,56,14,28},
		{0,19,38,53,15,28,41,58,30,13,56,43,17,2,55,36,60,47,26,9,51,32,21,6,34,49,4,23,45,62,11,24,59,40,29,14,52,39,18,1,37,54,3,16,42,57,12,31,7,20,33,50,8,27,46,61,25,10,63,44,22,5,48,35},
		{0,20,40,60,19,7,59,47,38,50,14,26,53,33,29,9,15,27,39,51,28,8,52,32,41,61,1,21,58,46,18,6,30,10,54,34,13,25,37,49,56,44,16,4,43,63,3,23,17,5,57,45,2,22,42,62,55,35,31,11,36,48,12,24},
		{0,21,42,63,23,2,61,40,46,59,4,17,57,44,19,6,31,10,53,32,8,29,34,55,49,36,27,14,38,51,12,25,62,43,20,1,41,60,3,22,16,5,58,47,7,18,45,56,33,52,11,30,54,35,28,9,15,26,37,48,24,13,50,39},
		{0,22,44,58,27,13,55,33,54,32,26,12,45,59,1,23,47,57,3,21,52,34,24,14,25,15,53,35,2,20,46,56,29,11,49,39,6,16,42,60,43,61,7,17,48,38,28,10,50,36,30,8,41,63,5,19,4,18,40,62,31,9,51,37},
		{0,23,46,57,31,8,49,38,62,41,16,7,33,54,15,24,63,40,17,6,32,55,14,25,1,22,47,56,30,9,48,39,61,42,19,4,34,53,12,27,3,20,45,58,28,11,50,37,2,21,44,59,29,10,51,36,60,43,18,5,35,52,13,26},
		{0,24,48,40,35,59,19,11,5,29,53,45,38,62,22,14,10,18,58,34,41,49,25,1,15,23,63,39,44,52,28,4,20,12,36,60,55,47,7,31,17,9,33,57,50,42,2,26,30,6,46,54,61,37,13,21,27,3,43,51,56,32,8,16},
		{0,25,50,43,39,62,21,12,13,20,63,38,42,51,24,1,26,3,40,49,61,36,15,22,23,14,37,60,48,41,2,27,52,45,6,31,19,10,33,56,57,32,11,18,30,7,44,53,46,55,28,5,9,16,59,34,35,58,17,8,4,29,54,47},
		{0,26,52,46,43,49,31,5,21,15,33,59,62,36,10,16,42,48,30,4,1,27,53,47,63,37,11,17,20,14,32,58,23,13,35,57,60,38,8,18,2,24,54,44,41,51,29,7,61,39,9,19,22,12,34,56,40,50,28,6,3,25,55,45},
		{0,27,54,45,47,52,25,2,29,6,43,48,50,41,4,31,58,33,12,23,21,14,35,56,39,60,17,10,8,19,62,37,55,44,1,26,24,3,46,53,42,49,28,7,5,30,51,40,13,22,59,32,34,57,20,15,16,11,38,61,63,36,9,18},
		{0,28,56,36,51,47,11,23,37,57,29,1,22,10,46,50,9,21,49,45,58,38,2,30,44,48,20,8,31,3,39,59,18,14,42,54,33,61,25,5,55,43,15,19,4,24,60,32,27,7,35,63,40,52,16,12,62,34,6,26,13,17,53,41},
		{0,29,58,39,55,42,13,16,45,48,23,10,26,7,32,61,25,4,35,62,46,51,20,9,52,41,14,19,3,30,57,36,50,47,8,21,5,24,63,34,31,2,37,56,40,53,18,15,43,54,17,12,28,1,38,59,6,27,60,33,49,44,11,22},
		{0,30,60,34,59,37,7,25,53,43,9,23,14,16,50,44,41,55,21,11,18,12,46,48,28,2,32,62,39,57,27,5,17,15,45,51,42,52,22,8,36,58,24,6,31,1,35,61,56,38,4,26,3,29,63,33,13,19,49,47,54,40,10,20},
		{0,31,62,33,63,32,1,30,61,34,3,28,2,29,60,35,57,38,7,24,6,25,56,39,4,27,58,37,59,36,5,26,49,46,15,16,14,17,48,47,12,19,50,45,51,44,13,18,8,23,54,41,55,40,9,22,53,42,11,20,10,21,52,43},
		{0,32,3,35,6,38,5,37,12,44,15,47,10,42,9,41,24,56,27,59,30,62,29,61,20,52,23,55,18,50,17,49,48,16,51,19,54,22,53,21,60,28,63,31,58,26,57,25,40,8,43,11,46,14,45,13,36,4,39,7,34,2,33,1},
		{0,33,1,32,2,35,3,34,4,37,5,36,6,39,7,38,8,41,9,40,10,43,11,42,12,45,13,44,14,47,15,46,16,49,17,48,18,51,19,50,20,53,21,52,22,55,23,54,24,57,25,56,26,59,27,58,28,61,29,60,30,63,31,62},
		{0,34,7,37,14,44,9,43,28,62,27,57,18,48,21,55,56,26,63,29,54,20,49,19,36,6,35,1,42,8,45,15,51,17,52,22,61,31,58,24,47,13,40,10,33,3,38,4,11,41,12,46,5,39,2,32,23,53,16,50,25,59,30,60},
		{0,35,5,38,10,41,15,44,20,55,17,50,30,61,27,56,40,11,45,14,34,1,39,4,60,31,57,26,54,21,51,16,19,48,22,53,25,58,28,63,7,36,2,33,13,46,8,43,59,24,62,29,49,18,52,23,47,12,42,9,37,6,32,3},
		{0,36,11,47,22,50,29,57,44,8,39,3,58,30,49,21,27,63,16,52,13,41,6,34,55,19,60,24,33,5,42,14,54,18,61,25,32,4,43,15,26,62,17,53,12,40,7,35,45,9,38,2,59,31,48,20,1,37,10,46,23,51,28,56},
		{0,37,9,44,18,55,27,62,36,1,45,8,54,19,63,26,11,46,2,39,25,60,16,53,47,10,38,3,61,24,52,17,22,51,31,58,4,33,13,40,50,23,59,30,32,5,41,12,29,56,20,49,15,42,6,35,57,28,48,21,43,14,34,7},
		{0,38,15,41,30,56,17,55,60,26,51,21,34,4,45,11,59,29,52,18,37,3,42,12,7,33,8,46,25,63,22,48,53,19,58,28,43,13,36,2,9,47,6,32,23,49,24,62,14,40,1,39,16,54,31,57,50,20,61,27,44,10,35,5},
		{0,39,13,42,26,61,23,48,52,19,57,30,46,9,35,4,43,12,38,1,49,22,60,27,31,56,18,53,5,34,8,47,21,50,24,63,15,40,2,37,33,6,44,11,59,28,54,17,62,25,51,20,36,3,41,14,10,45,7,32,16,55,29,58},
		{0,40,19,59,38,14,53,29,15,39,28,52,41,1,58,18,30,54,13,37,56,16,43,3,17,57,2,42,55,31,36,12,60,20,47,7,26,50,9,33,51,27,32,8,21,61,6,46,34,10,49,25,4,44,23,63,45,5,62,22,11,35,24,48},
		{0,41,17,56,34,11,51,26,7,46,22,63,37,12,52,29,14,39,31,54,44,5,61,20,9,32,24,49,43,2,58,19,28,53,13,36,62,23,47,6,27,50,10,35,57,16,40,1,18,59,3,42,48,25,33,8,21,60,4,45,55,30,38,15},
		{0,42,23,61,46,4,57,19,31,53,8,34,49,27,38,12,62,20,41,3,16,58,7,45,33,11,54,28,15,37,24,50,63,21,40,2,17,59,6,44,32,10,55,29,14,36,25,51,1,43,22,60,47,5,56,18,30,52,9,35,48,26,39,13},
		{0,43,21,62,42,1,63,20,23,60,2,41,61,22,40,3,46,5,59,16,4,47,17,58,57,18,44,7,19,56,6,45,31,52,10,33,53,30,32,11,8,35,29,54,34,9,55,28,49,26,36,15,27,48,14,37,38,13,51,24,12,39,25,50},
		{0,44,27,55,54,26,45,1,47,3,52,24,25,53,2,46,29,49,6,42,43,7,48,28,50,30,41,5,4,40,31,51,58,22,33,13,12,32,23,59,21,57,14,34,35,15,56,20,39,11,60,16,17,61,10,38,8,36,19,63,62,18,37,9},
		{0,45,25,52,50,31,43,6,39,10,62,19,21,56,12,33,13,32,20,57,63,18,38,11,42,7,51,30,24,53,1,44,26,55,3,46,40,5,49,28,61,16,36,9,15,34,22,59,23,58,14,35,37,8,60,17,48,29,41,4,2,47,27,54},
		{0,46,31,49,62,16,33,15,63,17,32,14,1,47,30,48,61,19,34,12,3,45,28,50,2,44,29,51,60,18,35,13,57,23,38,8,7,41,24,54,6,40,25,55,56,22,39,9,4,42,27,53,58,20,37,11,59,21,36,10,5,43,26,52},
		{0,47,29,50,58,21,39,8,55,24,42,5,13,34,16,63,45,2,48,31,23,56,10,37,26,53,7,40,32,15,61,18,25,54,4,43,35,12,62,17,46,1,51,28,20,59,9,38,52,27,41,6,14,33,19,60,3,44,30,49,57,22,36,11},
		{0,48,35,19,5,53,38,22,10,58,41,25,15,63,44,28,20,36,55,7,17,33,50,2,30,46,61,13,27,43,56,8,40,24,11,59,45,29,14,62,34,18,1,49,39,23,4,52,60,12,31,47,57,9,26,42,54,6,21,37,51,3,16,32},
		{0,49,33,16,1,48,32,17,2,51,35,18,3,50,34,19,4,53,37,20,5,52,36,21,6,55,39,22,7,54,38,23,8,57,41,24,9,56,40,25,10,59,43,26,11,58,42,27,12,61,45,28,13,60,44,29,14,63,47,30,15,62,46,31},
		{0,50,39,21,13,63,42,24,26,40,61,15,23,37,48,2,52,6,19,33,57,11,30,44,46,28,9,59,35,17,4,54,43,25,12,62,38,20,1,51,49,3,22,36,60,14,27,41,31,45,56,10,18,32,53,7,5,55,34,16,8,58,47,29},
		{0,51,37,22,9,58,44,31,18,33,55,4,27,40,62,13,36,23,1,50,45,30,8,59,54,5,19,32,63,12,26,41,11,56,46,29,2,49,39,20,25,42,60,15,16,35,53,6,47,28,10,57,38,21,3,48,61,14,24,43,52,7,17,34},
		{0,52,43,31,21,33,62,10,42,30,1,53,63,11,20,32,23,35,60,8,2,54,41,29,61,9,22,34,40,28,3,55,46,26,5,49,59,15,16,36,4,48,47,27,17,37,58,14,57,13,18,38,44,24,7,51,19,39,56,12,6,50,45,25},
		{0,53,41,28,17,36,56,13,34,23,11,62,51,6,26,47,7,50,46,27,22,35,63,10,37,16,12,57,52,1,29,40,14,59,39,18,31,42,54,3,44,25,5,48,61,8,20,33,9,60,32,21,24,45,49,4,43,30,2,55,58,15,19,38},
		{0,54,47,25,29,43,50,4,58,12,21,35,39,17,8,62,55,1,24,46,42,28,5,51,13,59,34,20,16,38,63,9,45,27,2,52,48,6,31,41,23,33,56,14,10,60,37,19,26,44,53,3,7,49,40,30,32,22,15,57,61,11,18,36},
		{0,55,45,26,25,46,52,3,50,5,31,40,43,28,6,49,39,16,10,61,62,9,19,36,21,34,56,15,12,59,33,22,13,58,32,23,20,35,57,14,63,8,18,37,38,17,11,60,42,29,7,48,51,4,30,41,24,47,53,2,1,54,44,27},
		{0,56,51,11,37,29,22,46,9,49,58,2,44,20,31,39,18,42,33,25,55,15,4,60,27,35,40,16,62,6,13,53,36,28,23,47,1,57,50,10,45,21,30,38,8,48,59,3,54,14,5,61,19,43,32,24,63,7,12,52,26,34,41,17},
		{0,57,49,8,33,24,16,41,1,56,48,9,32,25,17,40,2,59,51,10,35,26,18,43,3,58,50,11,34,27,19,42,4,61,53,12,37,28,20,45,5,60,52,13,36,29,21,44,6,63,55,14,39,30,22,47,7,62,54,15,38,31,23,46},
		{0,58,55,13,45,23,26,32,25,35,46,20,52,14,3,57,50,8,5,63,31,37,40,18,43,17,28,38,6,60,49,11,39,29,16,42,10,48,61,7,62,4,9,51,19,41,36,30,21,47,34,24,56,2,15,53,12,54,59,1,33,27,22,44},
		{0,59,53,14,41,18,28,39,17,42,36,31,56,3,13,54,34,25,23,44,11,48,62,5,51,8,6,61,26,33,47,20,7,60,50,9,46,21,27,32,22,45,35,24,63,4,10,49,37,30,16,43,12,55,57,2,52,15,1,58,29,38,40,19},
		{0,60,59,7,53,9,14,50,41,21,18,46,28,32,39,27,17,45,42,22,36,24,31,35,56,4,3,63,13,49,54,10,34,30,25,37,23,43,44,16,11,55,48,12,62,2,5,57,51,15,8,52,6,58,61,1,26,38,33,29,47,19,20,40},
		{0,61,57,4,49,12,8,53,33,28,24,37,16,45,41,20,1,60,56,5,48,13,9,52,32,29,25,36,17,44,40,21,2,63,59,6,51,14,10,55,35,30,26,39,18,47,43,22,3,62,58,7,50,15,11,54,34,31,27,38,19,46,42,23},
		{0,62,63,1,61,3,2,60,57,7,6,56,4,58,59,5,49,15,14,48,12,50,51,13,8,54,55,9,53,11,10,52,33,31,30,32,28,34,35,29,24,38,39,25,37,27,26,36,16,46,47,17,45,19,18,44,41,23,22,40,20,42,43,21},
		{0,63,61,2,57,6,4,59,49,14,12,51,8,55,53,10,33,30,28,35,24,39,37,26,16,47,45,18,41,22,20,43,1,62,60,3,56,7,5,58,48,15,13,50,9,54,52,11,32,31,29,34,25,38,36,27,17,46,44,19,40,23,21,42},
		};

		const unsigned char inv[64] = {0,1,33,62,49,43,31,44,57,37,52,28,46,40,22,25,61,54,51,39,26,35,14,24,23,15,20,34,11,53,45,6,63,2,27,21,56,9,50,19,13,47,48,5,7,30,12,41,42,4,38,18,10,29,17,60,36,8,59,58,55,16,3,32};
	#endif
	

	unsigned char w_row=row_weight[0];
	unsigned char w_col=col_weight[0];



	hipError_t err=hipSuccess;
    hipDeviceProp_t prop;
    
	uchar8 *cuda_ALPHAmn_=NULL;
	uchar8 *cuda_gamma=NULL;
	size_t size=sizeof(uchar8)*M*w_row;
	size_t size_cuda_gamma=sizeof(uchar8)*N;

    //allow pinned memory
    hipGetDeviceProperties(&prop, 0);
    if (prop.canMapHostMemory==0) 
        hipSetDeviceFlags(hipDeviceMapHost);

    //aloccate the image on pinned memory
    err=hipHostAlloc((void **)&cuda_ALPHAmn_,size, hipHostMallocDefault);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate cuda_ALPHAmn_ samples(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
	}
	
	err=hipHostAlloc((void **)&cuda_gamma,size_cuda_gamma, hipHostMallocDefault);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate cuda_gamma samples(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

	

	for (int index = 0; index < N; index++) {// initialize variable node message ALPHA with channel info GAMMA
		cuda_gamma[index].s0 = GAMMAn_[(Q*index)+0];
		cuda_gamma[index].s1 = GAMMAn_[(Q*index)+1];
		cuda_gamma[index].s2 = GAMMAn_[(Q*index)+2];
		cuda_gamma[index].s3 = GAMMAn_[(Q*index)+3];
		cuda_gamma[index].s4 = GAMMAn_[(Q*index)+4];
		cuda_gamma[index].s5 = GAMMAn_[(Q*index)+5];
		cuda_gamma[index].s6 = GAMMAn_[(Q*index)+6];
		cuda_gamma[index].s7 = GAMMAn_[(Q*index)+7];
	} 

	

	

	for (int index = 0; index < row_weight[0]*M; index++) {// initialize variable node message ALPHA with channel info GAMMA
		for (int a = 0; a < Q; a++) {
			ALPHAmn_[(index*Q)+a] = GAMMAn_[(Q*col_ind[index])+a];
			//printf("Alpha[%d][%d]=%d, col=%d\n", index,a,ALPHAmn_[(index*Q)+a],col_ind[index]);

		}
		//printf("row=%d, col=%d\n", 0, col_ind[index]);
	}

	for (int row = 0; row < M; row++) {
		for (int index = 0; index < row_weight[0]; index++) {// initialize variable node message ALPHA with channel info GAMMA
			for (int a = 0; a < Q; a++) {
				if(a==0){
					cuda_ALPHAmn_[(row*row_weight[0])+index].s0 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
					//if(col_ind[index]==1)
					//printf("Alpha[%d][%d]=%d, ol_ind[]=%d\n", index,a,cuda_ALPHAmn_[((index%16)*Q)+a].s0, col_ind[index]);
				}
				else if(a==1)
					cuda_ALPHAmn_[(row*row_weight[0])+index].s1 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
				else if(a==2)
					cuda_ALPHAmn_[(row*row_weight[0])+index].s2 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
				else if(a==3)
					cuda_ALPHAmn_[(row*row_weight[0])+index].s3 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
				else if(a==4)
					cuda_ALPHAmn_[(row*row_weight[0])+index].s4 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
				else if(a==5)
					cuda_ALPHAmn_[(row*row_weight[0])+index].s5 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
				else if(a==6)
					cuda_ALPHAmn_[(row*row_weight[0])+index].s6 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
				else if(a==7)
					cuda_ALPHAmn_[(row*row_weight[0])+index].s7 = ALPHAmn_[(((row*row_weight[0])+index)*Q)+a];
			}
		}
	}
	
	printf("w_row=%d\n",M);

	

	err=hipSuccess; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	//======================================================================================================================================================================
	//kernel dimensions
		#if Q==8
			dim3 FB_threadsPerBlock(128,1,1);
			dim3 FB_numBlocks(2,1,1);

			dim3 CN_threadsPerBlock(32,w_row,1);
			dim3 CN_numBlocks(8,1,1);

			dim3 VN_threadsPerBlock(w_col,8,1);
			dim3 VN_numBlocks(1,48,1);
		#elif Q==16
			dim3 FB_threadsPerBlock(8,1,Q);
			dim3 FB_numBlocks(2,1,1);

			dim3 CN_threadsPerBlock(4,w_row,Q);
			dim3 CN_numBlocks(4,1,1);

			dim3 VN_threadsPerBlock(w_col,32,Q);
			dim3 VN_numBlocks(1,12,1);
		#elif Q==32
			dim3 FB_threadsPerBlock(8,1,Q);
			dim3 FB_numBlocks(2,1,1);

			dim3 CN_threadsPerBlock(4,w_row,Q);
			dim3 CN_numBlocks(4,1,1);

			dim3 VN_threadsPerBlock(w_col,16,Q);
			dim3 VN_numBlocks(1,24,1);
		#endif
  
	//======================================================================================================================================================================
	//size of variables
		size_t size_gamma_8=sizeof(uchar8)*N;
		size_t size_alpha_8=sizeof(uchar8)*M*w_row;
		size_t size_FB_8=sizeof(uchar8)*M*w_row;
		size_t size_FB=sizeof(unsigned char)*M*w_row*Q;
		size_t size_dimension=sizeof(unsigned short);
		size_t size_GF=sizeof(unsigned char)*Q*Q;
		size_t size_GF_inv=sizeof(unsigned char)*Q;
		size_t size_row_ptr=sizeof(unsigned short)*(M+1);
		size_t size_val=sizeof(unsigned char)*M*w_row;
		size_t size_weight=sizeof(unsigned char);
		size_t size_col_ptr=sizeof(unsigned short)*(N+1);
		size_t size_ptr_val=sizeof(unsigned short)*M*w_row;
	
	//======================================================================================================================================================================
    //variables declaration
		uchar8 *d_gamma_8=NULL;
		uchar8 *d_alpha_8=NULL;;
		uchar8 *d_F_8=NULL;
		uchar8 *d_B_8=NULL;
		uchar8 *d_beta_8=NULL;

		uchar8 *h_alpha_8=NULL;
		uchar8 *h_F_8=NULL;
		uchar8 *h_B_8=NULL;
		uchar8 *h_beta_8=NULL;
		/* #if Q==32
			unsigned char *d_h_nb=NULL;
		#endif */
		
				
	//======================================================================================================================================================================
	//allocate host memory

		h_F=(unsigned char *)malloc(size_FB);
		if(h_F == NULL){
			printf("Failed to allocate host F\n" );
			exit(EXIT_FAILURE);
		}

		h_B=(unsigned char *)malloc(size_FB);
		if(h_B == NULL){
			printf("Failed to allocate host B\n" );
			exit(EXIT_FAILURE);
		}

		err=hipHostAlloc((void **)&h_F_8, size_FB_8, hipHostMallocDefault );
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate host h_F_8(error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipHostAlloc((void **)&h_B_8, size_FB_8, hipHostMallocDefault );
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate host h_B_8(error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipHostAlloc((void **)&h_alpha_8, size_alpha_8, hipHostMallocDefault );
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate host h_alpha_8(error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipHostAlloc((void **)&h_beta_8, size_alpha_8, hipHostMallocDefault );
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate host h_beta_8(error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		

	//======================================================================================================================================================================
    //allocate device memory
		err=hipMalloc((void **)&d_gamma_8, size_gamma_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device gamma (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_alpha_8, size_alpha_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device alpha8 (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_F_8, size_FB_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device F (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_B_8, size_FB_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device B (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMalloc((void **)&d_beta_8, size_alpha_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to allocate device beta (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}


		/* #if Q==32
			err=hipMalloc((void **)&d_h_nb, size_h_nb);
			if(err!=hipSuccess){
				fprintf(stderr, "Failed to allocate device h_nb (error code %d)!\n", hipGetLastError());
				exit(EXIT_FAILURE);
			}
		#endif */

	//======================================================================================================================================================================
    //device memory initialization
		err=hipMemset(d_gamma_8, 0, size_gamma_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device gamma (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_alpha_8, 0, size_alpha_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device alpha (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_F_8, 0, size_FB_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device d_F_8 (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_B_8, 0, size_FB_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device B (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemset(d_beta_8, 0, size_alpha_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to initialize device beta (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

				
	//======================================================================================================================================================================
	//copy data to constant

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_add), &add, size_GF,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy add from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_M), &M, size_dimension,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy M from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_N), &N, size_dimension,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy N from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_mult), &mult, size_GF,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy mult from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_inv), &inv, size_GF_inv,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy w_row from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_row_ptr), &row_ptr, size_row_ptr, 0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy row_ptr from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_val), &val, size_val,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy val from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_w_col), &w_col, size_weight,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy w_col from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_w_row), &w_row, size_weight,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy w_row from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_col_ptr), &col_ptr, size_col_ptr, 0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy row_ptr from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_ptr_to_val), &ptr_to_val, size_ptr_val,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy val from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_row_ind), &row_ind, size_ptr_val,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy row_ind from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpyToSymbol(HIP_SYMBOL(c_col_ind), &col_ind, size_ptr_val,0,hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy row_ind from host to constant (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}
	//======================================================================================================================================================================
	//copy data to device  

		err=hipMemcpy(d_gamma_8, cuda_gamma, size_gamma_8, hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy gamma from host to device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(d_alpha_8, cuda_ALPHAmn_, size_alpha_8, hipMemcpyHostToDevice);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy alpha_8 from host to device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}
	//======================================================================================================================================================================
	//execute the kernel
		hipEventRecord(start);
		for (int iter=0; iter < MAX_ITERATION; iter++) {
				GPU_FB_metrics<<<FB_numBlocks, FB_threadsPerBlock>>>( d_alpha_8, d_F_8, d_B_8, iter);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
				}

				GPU_CN<<<CN_numBlocks, CN_threadsPerBlock>>>( d_beta_8, d_F_8, d_B_8, iter);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE);
				} 

				GPU_VN<<<VN_numBlocks, VN_threadsPerBlock>>>( d_beta_8, d_alpha_8, d_gamma_8, iter);
				if(err!=hipSuccess){
					fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
					exit(EXIT_FAILURE); 
				} 
			//#endif
		}
		hipEventRecord(stop);


		

	//======================================================================================================================================================================
	//copy the data from device to host

		err=hipMemcpy(h_F_8, d_F_8, size_FB_8, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the F_8 from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(h_B_8, d_B_8, size_FB_8, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the B_8 from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(h_beta_8, d_beta_8, size_alpha_8, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the betta from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipMemcpy(cuda_ALPHAmn_, d_alpha_8, size_alpha_8, hipMemcpyDeviceToHost);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to copy the d_alpha_8 from device to host (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		hipEventSynchronize(stop);

	//======================================================================================================================================================================
	//free the device memory
		err=hipFree(d_gamma_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the gamma_8 from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_alpha_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the alpha_8 from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_F_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the F_8 from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_B_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the B_8 from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

		err=hipFree(d_beta_8);
		if(err!=hipSuccess){
			fprintf(stderr, "Failed to free the d_beta_8 from the device (error code %d)!\n", hipGetLastError());
			exit(EXIT_FAILURE);
		}

	//======================================================================================================================================================================
	//save data

		for (int row = 0; row < M; row++) {
			for (int index = 0; index < row_weight[0]; index++) {// initialize variable node message ALPHA with channel info GAMMA
				h_F[(row*Q*row_weight[0])+(0*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s0;
				h_F[(row*Q*row_weight[0])+(1*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s1;
				h_F[(row*Q*row_weight[0])+(2*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s2;
				h_F[(row*Q*row_weight[0])+(3*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s3;
				h_F[(row*Q*row_weight[0])+(4*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s4;
				h_F[(row*Q*row_weight[0])+(5*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s5;
				h_F[(row*Q*row_weight[0])+(6*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s6;
				h_F[(row*Q*row_weight[0])+(7*row_weight[0])+index]=h_F_8[(row*row_weight[0])+index].s7;
				
				h_B[(row*Q*row_weight[0])+(0*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s0;
				h_B[(row*Q*row_weight[0])+(1*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s1;
				h_B[(row*Q*row_weight[0])+(2*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s2;
				h_B[(row*Q*row_weight[0])+(3*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s3;
				h_B[(row*Q*row_weight[0])+(4*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s4;
				h_B[(row*Q*row_weight[0])+(5*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s5;
				h_B[(row*Q*row_weight[0])+(6*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s6;
				h_B[(row*Q*row_weight[0])+(7*row_weight[0])+index]=h_B_8[(row*row_weight[0])+index].s7;
			}
		}

		for (int row = 0; row < M; row++) {
			for (int index = 0; index < row_weight[0]; index++) {// initialize variable node message ALPHA with channel info GAMMA
				h_beta[(row*Q*row_weight[0])+(index*Q)+0]=h_beta_8[(row*row_weight[0])+index].s0;
				h_beta[(row*Q*row_weight[0])+(index*Q)+1]=h_beta_8[(row*row_weight[0])+index].s1;
				h_beta[(row*Q*row_weight[0])+(index*Q)+2]=h_beta_8[(row*row_weight[0])+index].s2;
				h_beta[(row*Q*row_weight[0])+(index*Q)+3]=h_beta_8[(row*row_weight[0])+index].s3;
				h_beta[(row*Q*row_weight[0])+(index*Q)+4]=h_beta_8[(row*row_weight[0])+index].s4;
				h_beta[(row*Q*row_weight[0])+(index*Q)+5]=h_beta_8[(row*row_weight[0])+index].s5;
				h_beta[(row*Q*row_weight[0])+(index*Q)+6]=h_beta_8[(row*row_weight[0])+index].s6;
				h_beta[(row*Q*row_weight[0])+(index*Q)+7]=h_beta_8[(row*row_weight[0])+index].s7;
			}
		}
		
		

		FILE *fp_B_val = fopen("./data/B_val.txt", "w");
		FILE *fp_F_val = fopen("./data/F_val.txt", "w");

		
		for (int row = 0; row < M; row++) {
			for (int a = 0; a < Q; a++) {	
				for (int col = 0; col < row_weight[0]; col++) {	//2. update BETA values from B and F values
					
					//if(row==0)
					//printf("F[%d][%d][%d]=%d\n",row, a,col,h_F[(row*Q*w_row)+(a*w_row)+col]);
					//printf("B[%d][%d][%d]=%d\n",row, a,w_row - index_c -1,h_B[(row*Q*w_row)+(a*w_row)+w_row - index_c -1]);
					//printf("Alpha[%d][%d][%d]=%d\n",row, col,a ,h_alpha[(Q*N*row)+(Q*col)+a]);
					fprintf(fp_B_val,"%d ",h_B[(row*Q*w_row)+(col*Q)+a]);
					fprintf(fp_F_val,"%d ", h_F[(row*Q*w_row)+(col*Q)+a]);
				}
				fprintf(fp_B_val,"\n");
				fprintf(fp_F_val,"\n");
			}
			fprintf(fp_B_val,"\n\n");
			fprintf(fp_F_val,"\n\n");
		}

		fclose(fp_B_val);
		fclose(fp_F_val);

	//======================================================================================================================================================================
	//free the host memory
		
	hipEventElapsedTime(&milliseconds, start, stop);

	/* #if Q==32
		free(h_nb);
	#endif */



	for (int row = 0; row < M; row++) {
		for (int index = 0; index < row_weight[0]; index++) {// initialize variable node message ALPHA with channel info GAMMA
			for (int a = 0; a < Q; a++) {
				if(a==0){
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s0 ;
					//if(col_ind[index]==1)
					//printf(index%16)*Q)+a].s0, col_ind[index])"Alpha[%d][%d]=%d, ol_ind[]=%d\n", index,a,cuda_ALPHAmn_[((;
				}
				else if(a==1)
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s1 ;
				else if(a==2)
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s2 ;
				else if(a==3)
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s3 ;
				else if(a==4)
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s4 ;
				else if(a==5)
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s5 ;
				else if(a==6)
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s6 ;
				else if(a==7)
					ALPHAmn_[(((row*row_weight[0])+index)*Q)+a]=cuda_ALPHAmn_[(row*row_weight[0])+index].s7 ;
			}

		}
	}
    
    err=hipHostFree(cuda_ALPHAmn_);
	if(err!=hipSuccess){
		fprintf(stderr, "Failed to free the cuda_ALPHAmn_ from the host (error code %d)!\n", hipGetLastError());
		exit(EXIT_FAILURE);
	}

	err=hipHostFree(cuda_gamma);
	if(err!=hipSuccess){
		fprintf(stderr, "Failed to free the cuda_gamma from the host (error code %d)!\n", hipGetLastError());
		exit(EXIT_FAILURE);
	}

	err=hipHostFree(h_B_8);
	if(err!=hipSuccess){
		fprintf(stderr, "Failed to free the h_B_8 from the host (error code %d)!\n", hipGetLastError());
		exit(EXIT_FAILURE);
	}

	err=hipHostFree(h_F_8);
	if(err!=hipSuccess){
		fprintf(stderr, "Failed to free the h_F_8 from the host (error code %d)!\n", hipGetLastError());
		exit(EXIT_FAILURE);
	}

	err=hipHostFree(h_beta_8);
	if(err!=hipSuccess){
		fprintf(stderr, "Failed to free the h_beta_8 from the host (error code %d)!\n", hipGetLastError());
		exit(EXIT_FAILURE);
	}

	
	

			
	return 0;
}
